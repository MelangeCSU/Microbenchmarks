
#include <hip/hip_runtime.h>
/*Author: Swetha Varadarajan 6/10/2015
 Adapted from Waruna Ranasinghe's summer 2014 work on kpdp micro-benchmarking 
 Code generated from intmazaddF5CG.c 
 Parameters list
size of A = 960
    size of B = 90
    gridDim = 30
    blockDim = 320
    Iterations k= 1700000
   Values per thread x = 3
*/

__global__ void IntmaxaddF5(const int* A, const int* B, int* C, int x, int k)
{
	int B_start_index = (blockIdx.x*gridDim.y + blockIdx.y)*x;
	int A_start_index = (threadIdx.x*blockDim.y + threadIdx.y)*x;
	int C_width = x*gridDim.x*gridDim.y;


	int i,j,t;
	__shared__ int c[320][3][3];
	int a_0, a_1, a_2;
	int b_0, b_1, b_2;


	a_0 = A[A_start_index+0];
	a_1 = A[A_start_index+1];
	a_2 = A[A_start_index+2];


	b_0 = B[B_start_index+0];
	b_1 = B[B_start_index+1];
	b_2 = B[B_start_index+2];


	#pragma unroll
	for(i=0;i<x;i++)
	{
		for(j=0;j<x;j++)
		{
			c[threadIdx.x][i][j]=0;
		}
	}
	for (t = 0; t < k; t++) {
		c[threadIdx.x][0][0] = max(a_0,(c[threadIdx.x][2][2]+b_0));
		c[threadIdx.x][0][1] = max(a_0,(c[threadIdx.x][2][1]+b_1));
		c[threadIdx.x][0][2] = max(a_0,(c[threadIdx.x][2][0]+b_2));
		c[threadIdx.x][1][0] = max(a_1,(c[threadIdx.x][1][2]+b_0));
		c[threadIdx.x][1][1] = max(a_1,(c[threadIdx.x][1][1]+b_1));
		c[threadIdx.x][1][2] = max(a_1,(c[threadIdx.x][1][0]+b_2));
		c[threadIdx.x][2][0] = max(a_2,(c[threadIdx.x][0][2]+b_0));
		c[threadIdx.x][2][1] = max(a_2,(c[threadIdx.x][0][1]+b_1));
		c[threadIdx.x][2][2] = max(a_2,(c[threadIdx.x][0][0]+b_2));


		a_0 = max(a_0,(b_0+c[threadIdx.x][0][0]));
		a_1 = max(a_1,(b_1+c[threadIdx.x][0][1]));
		a_2 = max(a_2,(b_2+c[threadIdx.x][0][2]));


		b_0 =  max(b_0,(a_0+c[threadIdx.x][1][0]));
		b_1 =  max(b_1,(a_1+c[threadIdx.x][1][1]));
		b_2 =  max(b_2,(a_2+c[threadIdx.x][1][2]));


	}


	C[(A_start_index+0)*C_width + B_start_index+0] = c[threadIdx.x][0][0];
	C[(A_start_index+0)*C_width + B_start_index+1] = c[threadIdx.x][0][1];
	C[(A_start_index+0)*C_width + B_start_index+2] = c[threadIdx.x][0][2];
	C[(A_start_index+1)*C_width + B_start_index+0] = c[threadIdx.x][1][0];
	C[(A_start_index+1)*C_width + B_start_index+1] = c[threadIdx.x][1][1];
	C[(A_start_index+1)*C_width + B_start_index+2] = c[threadIdx.x][1][2];
	C[(A_start_index+2)*C_width + B_start_index+0] = c[threadIdx.x][2][0];
	C[(A_start_index+2)*C_width + B_start_index+1] = c[threadIdx.x][2][1];
	C[(A_start_index+2)*C_width + B_start_index+2] = c[threadIdx.x][2][2];


}
