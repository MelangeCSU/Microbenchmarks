
#include <hip/hip_runtime.h>
__global__ void IntmaxaddT4(int* C, int k, int sl, int sharedsize)
{


	int i,t;
	int a=0,a2=0,b=0;
	int blockStartIndex  = blockIdx.x * blockDim.x * sharedsize;
	int threadStartIndex = blockStartIndex + threadIdx.x ;
	__shared__ int c[4000];


	#pragma unroll
	for(i = threadStartIndex; i < sharedsize; i+=sl)
		c[threadIdx.x+i]=C[threadIdx.x+i];

	for (t = 0; t < k; t++) {
		
		#pragma unroll
		for (i = threadStartIndex; i < sharedsize; i+=sl){
			a2=c[i];
			c[i] = max(a,(c[i]+b));
			a=a2;
		a=max(a,c[threadIdx.x]+10);
	}}

	#pragma unroll
	for(i = threadStartIndex; i < sharedsize; i+=sl)
		C[threadIdx.x+i]=c[threadIdx.x+i];



}
