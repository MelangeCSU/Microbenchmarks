
#include <hip/hip_runtime.h>
__global__ void IntmaxaddT4(int* C, int k, int sl)
{


	int i,t;
	int a=0,a2=0,b=2;
	int sharedsize=234;
	int blockStartIndex  = blockIdx.x * blockDim.x * 234;
	int threadStartIndex = blockStartIndex + threadIdx.x ;
	__shared__ int c[234];


	#pragma unroll
	for(i = threadStartIndex; i < sharedsize; i+=sl)
		c[threadIdx.x+i]=C[threadIdx.x+i];

	for (t = 0; t < k; t++) {
		a=max(a,c[threadIdx.x]+10);
		#pragma unroll
		for (i = threadStartIndex; i < sharedsize; i+=sl){
		a2=c[i];c[i] = max(a,(c[i]+b));a=a2;
	}}

	#pragma unroll
	for(i = threadStartIndex; i < sharedsize; i+=sl)
		C[threadIdx.x+i]=c[threadIdx.x+i];



}
