#include "hip/hip_runtime.h"
/*Author: Swetha Varadarajan 6/10/2015
Adapted from CSU CS575 Spring 2011 
	which was based on code from the CUDA Programming Guide by David Newman
	Modified by Wim Bohm and David Newman
Changes made:
	1. Replaced cutil.h funcitons with normal timer functions adapted from CS475 assignments 
	2. Included the extern "C" in order to compile C code alongwith nvcc. 
	3. Replaced the Kernel and the header file name according to the functionality. 
	4. Kernel functionality changed from vector multiplication to max-add.	
	5. Time calculated is in seconds. So, the corresponding calculations are modified. (GOPS,GBYTES,TIME)
	6. FLOPS changed to OPS since this is an integer operation.
	7. Floats to Ints data type change for host and device vectors. 
*/

#include <stdio.h>
#include "intmaxaddT4K.h"
#include <math.h>

extern "C" {
#include "timer.h"
}

// Variables for host and device vectors.
//int* h_A; 
//int* h_B; 
int* h_C; 
//int* d_A; 
//int* d_B; 
int* d_C; 

// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{
    	
   	//int ValuesPerThread; //Values per thread
	int k; // no. of repeatitions
	int sl;//stride length
	int gridWidth = 60;
	int blockWidth = 1;

	// Parse arguments.
    	if(argc != 5){
    		 printf("Usage: %s Values per thread Iterations\n", argv[0]);
     		// printf("Stride-length is the sweep radius of each thread.\n");
    		 printf("Total vector size is 128 * 60 * this value.\n");
    		 printf("Iterations is the number of repeatitions done by each thread.\n");
    		 exit(0);
   		 } 
	else 	{
     		 sscanf(argv[1], "%d", &sl);
      		
		 sscanf(argv[2], "%d", &k);
       		 sscanf(argv[3], "%d", &gridWidth);
      		 sscanf(argv[4], "%d", &blockWidth);
		// sscanf(argv[5], "%d", &sl);
		 
    		}      
	//maximum shared memory size * # SMs * TPB
	// shared memory size =  16384 bytes = 4096 ints (theoretical)
	// Practical it is 4000 ints. 
	// 4000*30*threads per thread block
	int size_C = 4000* 30 * blockWidth;
        //int size_C = ValuesPerThread * blockWidth * gridWidth;
	
    	dim3 dimGrid(gridWidth);                    
    	dim3 dimBlock(blockWidth);                 

	
	h_C = (int*)malloc(size_C*sizeof(int));
    	if (h_C == 0) Cleanup(false);

   	// Allocate vectors in device memory.
   	hipError_t error;

	error = hipMalloc((void**)&d_C, size_C*sizeof(int));
    	if (error != hipSuccess) Cleanup(false);


   	IntmaxaddT4<<<dimGrid, dimBlock>>>(d_C, k , sl);
    	error = hipGetLastError();

  	if (error != hipSuccess) {
		printf("W: %s\n", hipGetErrorString(error));
		Cleanup(false);
	}
    	hipDeviceSynchronize();



    	// Initialize timer
   	double time;
   	initialize_timer();
  	start_timer();
  
    	// Invoke kernel
    	IntmaxaddT4<<<dimGrid, dimBlock>>>( d_C, k , sl);
    	error = hipGetLastError();
    	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
		Cleanup(false);
	}


	// Compute elapsed time 
   	hipDeviceSynchronize();
	stop_timer();
   	time = elapsed_time();
	int cal = floor((120000/(gridWidth*sl)))+1;
	//printf("cal=%d\n",cal);
      // double nops = (double)size_A*(double)size_B*(double)k*(double)2 + (double)4*(double)k*(double)ValuesPerThread*(double)gridWidth*(double)blockWidth;
	double nops = (double)cal*(double)2*(double)k*(double)blockWidth*(double)gridWidth;
	//printf("cal=%d\tnops=%lf\n",cal,nops);
        float nopsPerSec = float(nops)/time;
    	float nGopsPerSec = nopsPerSec*1e-9;
	
	// Compute transfer rates.
    //	int nBytes = size_A*sizeof(int) + size_B*sizeof(int) + size_A*sizeof(int)*size_B*sizeof(int); // 2N words in, N*N word out
	//int nBytes = size_A*sizeof(int) + size_B*sizeof(int) + size_A*sizeof(int)*size_B*sizeof(int); // 2N words in, N*N word out
    //	float nBytesPerSec = (float)nBytes/time;
    	//float nGBytesPerSec = nBytesPerSec*1e-9;

	// Report timing data.*/
    	//printf( "Time: %f (s), Gops: %f, GBytesS: %f\n", time, nGopsPerSec, nGBytesPerSec);
	printf( "Time: %f (s), Gops: %f\n", time, nGopsPerSec);
    // printf( "Time: %f (s) \n", time);
    	// Copy result from device memory to host memory
   	error = hipMemcpy(h_C, d_C, size_C*sizeof(int), hipMemcpyDeviceToHost);
    	if (error != hipSuccess) Cleanup(false);


    	Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
  
    if (d_C)
        hipFree(d_C);

  
    if (h_C)
        free(h_C);
        
    error = hipDeviceReset();
    checkCUDAError("tmp"); 
    if (!noError || error != hipSuccess)
        printf("error: %s cuda malloc or cuda thread exit failed \n", hipGetErrorString(hipGetLastError()));
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
		printf("Error");
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      exit(-1);
    }                         
}


