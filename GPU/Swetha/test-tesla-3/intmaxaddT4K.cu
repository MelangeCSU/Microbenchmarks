
#include <hip/hip_runtime.h>
__global__ void IntmaxaddT4(int* C, int k, int sl)
{


	int i,t;
	int a=0,a2=0,b=2;
	int N=120000/gridDim.x;
	int blockStartIndex  = blockIdx.x * N;
	int threadStartIndex = blockStartIndex + threadIdx.x ;
	int threadEndIndex   = blockStartIndex + N;
	__shared__ int c[500];//array count=N


	int count=0;
	#pragma unroll
	for(i = threadStartIndex; i < threadEndIndex; i+=sl){
		c[threadIdx.x+count]=C[threadIdx.x+i];
	count+=sl;}
	for (t = 0; t < k; t++) {
		a=max(a,c[threadIdx.x]+10);
		#pragma unroll
		for(i = threadIdx.x; i < (threadIdx.x+count); i+=sl){
		a2=c[i];c[i] = max(a,(c[i]+b));a=a2;
	}}
	count=0;

	#pragma unroll
	for(i = threadStartIndex; i < threadEndIndex; i+=sl){
		C[threadIdx.x+i]=c[threadIdx.x+count];
	count++;}


}
