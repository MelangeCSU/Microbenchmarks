
#include <hip/hip_runtime.h>
/***********************************
BW Benchmark :s += A * B
size of A,B : 196608
TILE_SIZE     : 6144

************************************/

#define TILE_SIZE 6144
#define SHARED_SIZE 6144

__global__ void BWkernel (int *C, int x)
{
	 int tId = threadIdx.x;
	 int bDim = blockDim.x*1;


	int i, t, s[1];
	
	__shared__ int s_A[SHARED_SIZE], s_B[SHARED_SIZE];

	//Main loop
	#pragma unroll 1
	for(t=0; t<4000000; t++)
	{
	#pragma unroll 1
	for(i=0; i<TILE_SIZE; i+=bDim)
	{
		s[0] += s_A[i+tId+0] * s_B[i+tId+0];
	
	}
	}
	
	//Store result to global memory	
if(tId<1)		C[tId] = s[tId];
	
}
