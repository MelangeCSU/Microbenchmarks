
#include <hip/hip_runtime.h>
/***********************************
BW Benchmark :C += A * B
size of A,B,C : 196608
TILE_SIZE     : 6144

************************************/

#define TILE_SIZE 6144
#define SHARED_SIZE 6144

__global__ void BWkernel (int *C, int x)
{
	 int tId = threadIdx.x;
	 int bDim = blockDim.x;


	int i, t;
	
	__shared__ int s_A[SHARED_SIZE], s_C[SHARED_SIZE];

	//Main loop
	#pragma unroll 1
	for(t=0; t<7000000; t++)
	{
	#pragma unroll 1
	for(i=0; i<TILE_SIZE; i+=bDim)
	{
		s_C[i+tId] = s_A[i+tId];
	
	}
	}
	
	//Store result to global memory	
		C[tId] = s_C[tId];
	
}
