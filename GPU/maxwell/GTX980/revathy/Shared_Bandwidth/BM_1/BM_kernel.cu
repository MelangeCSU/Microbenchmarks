
#include <hip/hip_runtime.h>
/***********************************
BW Benchmark :C += A * B
size of A,B,C : 131072
TILE_SIZE     : 4096

************************************/

#define TILE_SIZE 4096
#define SHARED_SIZE 4096

__global__ void BWkernel (int *C, int x)
{
	 int tId = threadIdx.x;
	 int bDim = blockDim.x;


	int i, t;
	
	__shared__ int s_A[SHARED_SIZE], s_B[SHARED_SIZE], s_C[SHARED_SIZE];

	//Main loop
	#pragma unroll 1
	for(t=0; t<4000000; t++)
	{
	#pragma unroll 8
	for(i=0; i<TILE_SIZE; i+=bDim)
	{
		s_C[i+tId] += s_A[i+tId] * s_B[i+tId];
	
	}
	}
	
	//Store result to global memory	
		C[tId] = s_C[tId];
	
}
