
#include <hip/hip_runtime.h>
/***********************************
BW Benchmark :C = A + B
size of A,B,C : 32768
TILE_SIZE     : 1024

************************************/

#define TILE_SIZE 1024
#define SHARED_SIZE 4096

__global__ void BWkernel (int *C, int x)
{
	 int tId = threadIdx.x;
	 int bDim = blockDim.x;


	int i, t;
	
	__shared__ int s_A[SHARED_SIZE], s_B[SHARED_SIZE], s_C[SHARED_SIZE];

	//Main loop
	#pragma unroll 1
	for(t=0; t<4000000; t++)
	{
	#pragma unroll 2
	for(i=0; i<TILE_SIZE; i+=bDim)
	{
		s_C[i+tId] = s_A[i+tId] + s_B[i+tId];
	
	}
	}
	
	//Store result to global memory	
		C[tId] = s_C[tId];
	
}
