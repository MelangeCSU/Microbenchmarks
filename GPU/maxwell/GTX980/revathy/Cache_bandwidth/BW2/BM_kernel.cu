
#include <hip/hip_runtime.h>
/***********************************
This kernel is for testing L1 and L2 cache Bandwidth

BW Benchmark :C[i] = A[i]+B[i]
TILE_SIZE     : 1024

************************************/

#define TILE_SIZE 1024

__global__ void BWkernel (int *A, int *B, int *C, int x)
{
	 int tId = threadIdx.x;
	 int bId = blockIdx.x;
	 int bDim = blockDim.x;

	int start_Idx = bId*TILE_SIZE + tId;

	int i, s;
	

	s = 0;

	//Main loop
	#pragma unroll
	for(i=start_Idx; i<TILE_SIZE; i+=bDim)
	{
		C[i] = A[i] + B[i];
	
	}
	
	if(tId==0) C[bId] = s; 
}
