
#include <hip/hip_runtime.h>
/** size of A = 640
    size of B = 300
    gridDim = 60
    blockDim = 128
    k= 200000
    x = 5
**/

__global__ void CompareAddVectors(const int* A, const int* B, int* C, int x, int k)
{
	int size_A = x*blockDim.x;
	int B_start_index = (blockIdx.x*gridDim.y + blockIdx.y)*x;

	int t,i,j,temp;
	__shared__ int c[5][640]; 
	 __shared__ int a[640]; 
	int b_0, b_1, b_2, b_3, b_4;


	for (t = 0; t < x; t++) {
		temp = blockDim.x*t + threadIdx.x;
		a[temp] = A[temp];
	}


	for (i = 0; i < x; i++) {
		for (t = 0; t < x; t++) {
			c[i][blockDim.x*t + threadIdx.x] = 0;
		}
	}


	__syncthreads(); 

	b_0 = B[B_start_index+0];
	b_1 = B[B_start_index+1];
	b_2 = B[B_start_index+2];
	b_3 = B[B_start_index+3];
	b_4 = B[B_start_index+4];




	for (t = 0; t < k; t++) {
		#pragma unroll
		for (j = 0; j < x; j++) {
			temp = blockDim.x*j+threadIdx.x;
			c[0][temp] = max(a[temp]+b_0, c[0][temp]);
			c[1][temp] = max(a[temp]+b_1, c[1][temp]);
			c[2][temp] = max(a[temp]+b_2, c[2][temp]);
			c[3][temp] = max(a[temp]+b_3, c[3][temp]);
			c[4][temp] = max(a[temp]+b_4, c[4][temp]);
		}


		for (i = 0; i < x; i++) {
			temp = blockDim.x*i+threadIdx.x;
			a[temp] = max(a[temp]+10, a[temp]);
		}


		b_0 = max(b_0, b_0+10);
		b_1 = max(b_1, b_1+10);
		b_2 = max(b_2, b_2+10);
		b_3 = max(b_3, b_3+10);
		b_4 = max(b_4, b_4+10);


		__syncthreads();
	}


	for (i = 0; i < x; i++) {
		for (t = 0; t < x; t++) {
			temp = t*blockDim.x + threadIdx.x;
			C[(x*blockIdx.x+i)*size_A + temp] = c[i][temp];
		}
	}


}
