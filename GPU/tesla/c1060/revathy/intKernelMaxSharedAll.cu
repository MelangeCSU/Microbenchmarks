
#include <hip/hip_runtime.h>
/** size of A = 8
    size of B = 16
    gridDim = 4
    blockDim = 2
    k= 2
    x = 4
**/

__global__ void CompareAddVectors(const int* A, const int* B, int* C, int x, int k)
{
	int size_A = x*blockDim.x;
	int B_start_index = (blockIdx.x*gridDim.y + blockIdx.y)*x;

	int t,i,j,temp;
	__shared__ int c[4][8]; 
	 __shared__ int a[8]; 
	int b_0, b_1, b_2, b_3;


	for (t = 0; t < 4; t++) {
		temp = blockDim.x*t + threadIdx.x;
		a[temp] = A[temp];
	}


	for (i = 0; i < 4; i++) {
		for (t = 0; t < 4; t++) {
			c[i][blockDim.x*t + threadIdx.x] = 0;
		}
	}


	__syncthreads(); 

	b_0 = B[B_start_index+0];
	b_1 = B[B_start_index+1];
	b_2 = B[B_start_index+2];
	b_3 = B[B_start_index+3];




	#pragma unroll
	for (t = 0; t < 2; t++) {
		temp = blockDim.x*0+threadIdx.x;
		c[0][temp] = max(a[temp]+b_0, c[0][temp]);
		temp = blockDim.x*1+threadIdx.x;
		c[0][temp] = max(a[temp]+b_0, c[0][temp]);
		temp = blockDim.x*2+threadIdx.x;
		c[0][temp] = max(a[temp]+b_0, c[0][temp]);
		temp = blockDim.x*3+threadIdx.x;
		c[0][temp] = max(a[temp]+b_0, c[0][temp]);
		temp = blockDim.x*0+threadIdx.x;
		c[1][temp] = max(a[temp]+b_1, c[1][temp]);
		temp = blockDim.x*1+threadIdx.x;
		c[1][temp] = max(a[temp]+b_1, c[1][temp]);
		temp = blockDim.x*2+threadIdx.x;
		c[1][temp] = max(a[temp]+b_1, c[1][temp]);
		temp = blockDim.x*3+threadIdx.x;
		c[1][temp] = max(a[temp]+b_1, c[1][temp]);
		temp = blockDim.x*0+threadIdx.x;
		c[2][temp] = max(a[temp]+b_2, c[2][temp]);
		temp = blockDim.x*1+threadIdx.x;
		c[2][temp] = max(a[temp]+b_2, c[2][temp]);
		temp = blockDim.x*2+threadIdx.x;
		c[2][temp] = max(a[temp]+b_2, c[2][temp]);
		temp = blockDim.x*3+threadIdx.x;
		c[2][temp] = max(a[temp]+b_2, c[2][temp]);
		temp = blockDim.x*0+threadIdx.x;
		c[3][temp] = max(a[temp]+b_3, c[3][temp]);
		temp = blockDim.x*1+threadIdx.x;
		c[3][temp] = max(a[temp]+b_3, c[3][temp]);
		temp = blockDim.x*2+threadIdx.x;
		c[3][temp] = max(a[temp]+b_3, c[3][temp]);
		temp = blockDim.x*3+threadIdx.x;
		c[3][temp] = max(a[temp]+b_3, c[3][temp]);


		temp = blockDim.x*0+threadIdx.x;
		a[temp] = max(a[temp]+10, a[temp]);
		temp = blockDim.x*1+threadIdx.x;
		a[temp] = max(a[temp]+10, a[temp]);
		temp = blockDim.x*2+threadIdx.x;
		a[temp] = max(a[temp]+10, a[temp]);
		temp = blockDim.x*3+threadIdx.x;
		a[temp] = max(a[temp]+10, a[temp]);


		b_0 = max(b_0, b_0+10);
		b_1 = max(b_1, b_1+10);
		b_2 = max(b_2, b_2+10);
		b_3 = max(b_3, b_3+10);


		__syncthreads();
	}


	for (i = 0; i < x; i++) {
		for (t = 0; t < x; t++) {
			temp = t*blockDim.x + threadIdx.x;
			C[(x*blockIdx.x+i)*size_A + temp] = c[i][temp];
		}
	}


}
