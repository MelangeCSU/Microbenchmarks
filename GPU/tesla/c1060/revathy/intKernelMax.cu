
#include <hip/hip_runtime.h>
/** This file has been generated by CodeGenInt_max.c
    size of A = 576
    size of B = 540
    gridDim = 60
    blockDim = 64
    k= 500000
    x = 9
**/

__global__ void CompareAddVectors(const int* A, const int* B, int* C, int x, int k)
{
	int B_start_index = (blockIdx.x*gridDim.y + blockIdx.y)*x;
	int A_start_index = (threadIdx.x*blockDim.y + threadIdx.y)*x;
	int C_width = x*gridDim.x*gridDim.y;


	int t;
	int c_0_0, c_0_1, c_0_2, c_0_3, c_0_4, c_0_5, c_0_6, c_0_7, c_0_8, c_1_0, c_1_1, c_1_2, c_1_3, c_1_4, c_1_5, c_1_6, c_1_7, c_1_8, c_2_0, c_2_1, c_2_2, c_2_3, c_2_4, c_2_5, c_2_6, c_2_7, c_2_8, c_3_0, c_3_1, c_3_2, c_3_3, c_3_4, c_3_5, c_3_6, c_3_7, c_3_8, c_4_0, c_4_1, c_4_2, c_4_3, c_4_4, c_4_5, c_4_6, c_4_7, c_4_8, c_5_0, c_5_1, c_5_2, c_5_3, c_5_4, c_5_5, c_5_6, c_5_7, c_5_8, c_6_0, c_6_1, c_6_2, c_6_3, c_6_4, c_6_5, c_6_6, c_6_7, c_6_8, c_7_0, c_7_1, c_7_2, c_7_3, c_7_4, c_7_5, c_7_6, c_7_7, c_7_8, c_8_0, c_8_1, c_8_2, c_8_3, c_8_4, c_8_5, c_8_6, c_8_7, c_8_8;
	int a_0, a_1, a_2, a_3, a_4, a_5, a_6, a_7, a_8;
	int b_0, b_1, b_2, b_3, b_4, b_5, b_6, b_7, b_8;


	a_0 = A[A_start_index+0];
	a_1 = A[A_start_index+1];
	a_2 = A[A_start_index+2];
	a_3 = A[A_start_index+3];
	a_4 = A[A_start_index+4];
	a_5 = A[A_start_index+5];
	a_6 = A[A_start_index+6];
	a_7 = A[A_start_index+7];
	a_8 = A[A_start_index+8];


	b_0 = B[B_start_index+0];
	b_1 = B[B_start_index+1];
	b_2 = B[B_start_index+2];
	b_3 = B[B_start_index+3];
	b_4 = B[B_start_index+4];
	b_5 = B[B_start_index+5];
	b_6 = B[B_start_index+6];
	b_7 = B[B_start_index+7];
	b_8 = B[B_start_index+8];


	c_0_0 = 0;
	c_0_1 = 0;
	c_0_2 = 0;
	c_0_3 = 0;
	c_0_4 = 0;
	c_0_5 = 0;
	c_0_6 = 0;
	c_0_7 = 0;
	c_0_8 = 0;
	c_1_0 = 0;
	c_1_1 = 0;
	c_1_2 = 0;
	c_1_3 = 0;
	c_1_4 = 0;
	c_1_5 = 0;
	c_1_6 = 0;
	c_1_7 = 0;
	c_1_8 = 0;
	c_2_0 = 0;
	c_2_1 = 0;
	c_2_2 = 0;
	c_2_3 = 0;
	c_2_4 = 0;
	c_2_5 = 0;
	c_2_6 = 0;
	c_2_7 = 0;
	c_2_8 = 0;
	c_3_0 = 0;
	c_3_1 = 0;
	c_3_2 = 0;
	c_3_3 = 0;
	c_3_4 = 0;
	c_3_5 = 0;
	c_3_6 = 0;
	c_3_7 = 0;
	c_3_8 = 0;
	c_4_0 = 0;
	c_4_1 = 0;
	c_4_2 = 0;
	c_4_3 = 0;
	c_4_4 = 0;
	c_4_5 = 0;
	c_4_6 = 0;
	c_4_7 = 0;
	c_4_8 = 0;
	c_5_0 = 0;
	c_5_1 = 0;
	c_5_2 = 0;
	c_5_3 = 0;
	c_5_4 = 0;
	c_5_5 = 0;
	c_5_6 = 0;
	c_5_7 = 0;
	c_5_8 = 0;
	c_6_0 = 0;
	c_6_1 = 0;
	c_6_2 = 0;
	c_6_3 = 0;
	c_6_4 = 0;
	c_6_5 = 0;
	c_6_6 = 0;
	c_6_7 = 0;
	c_6_8 = 0;
	c_7_0 = 0;
	c_7_1 = 0;
	c_7_2 = 0;
	c_7_3 = 0;
	c_7_4 = 0;
	c_7_5 = 0;
	c_7_6 = 0;
	c_7_7 = 0;
	c_7_8 = 0;
	c_8_0 = 0;
	c_8_1 = 0;
	c_8_2 = 0;
	c_8_3 = 0;
	c_8_4 = 0;
	c_8_5 = 0;
	c_8_6 = 0;
	c_8_7 = 0;
	c_8_8 = 0;


#pragma unroll
	for (t = 0; t < 500000; t++) {
		c_0_0 = max(c_0_0,a_0+b_0);
		c_0_1 = max(c_0_1,a_0+b_1);
		c_0_2 = max(c_0_2,a_0+b_2);
		c_0_3 = max(c_0_3,a_0+b_3);
		c_0_4 = max(c_0_4,a_0+b_4);
		c_0_5 = max(c_0_5,a_0+b_5);
		c_0_6 = max(c_0_6,a_0+b_6);
		c_0_7 = max(c_0_7,a_0+b_7);
		c_0_8 = max(c_0_8,a_0+b_8);
		c_1_0 = max(c_1_0,a_1+b_0);
		c_1_1 = max(c_1_1,a_1+b_1);
		c_1_2 = max(c_1_2,a_1+b_2);
		c_1_3 = max(c_1_3,a_1+b_3);
		c_1_4 = max(c_1_4,a_1+b_4);
		c_1_5 = max(c_1_5,a_1+b_5);
		c_1_6 = max(c_1_6,a_1+b_6);
		c_1_7 = max(c_1_7,a_1+b_7);
		c_1_8 = max(c_1_8,a_1+b_8);
		c_2_0 = max(c_2_0,a_2+b_0);
		c_2_1 = max(c_2_1,a_2+b_1);
		c_2_2 = max(c_2_2,a_2+b_2);
		c_2_3 = max(c_2_3,a_2+b_3);
		c_2_4 = max(c_2_4,a_2+b_4);
		c_2_5 = max(c_2_5,a_2+b_5);
		c_2_6 = max(c_2_6,a_2+b_6);
		c_2_7 = max(c_2_7,a_2+b_7);
		c_2_8 = max(c_2_8,a_2+b_8);
		c_3_0 = max(c_3_0,a_3+b_0);
		c_3_1 = max(c_3_1,a_3+b_1);
		c_3_2 = max(c_3_2,a_3+b_2);
		c_3_3 = max(c_3_3,a_3+b_3);
		c_3_4 = max(c_3_4,a_3+b_4);
		c_3_5 = max(c_3_5,a_3+b_5);
		c_3_6 = max(c_3_6,a_3+b_6);
		c_3_7 = max(c_3_7,a_3+b_7);
		c_3_8 = max(c_3_8,a_3+b_8);
		c_4_0 = max(c_4_0,a_4+b_0);
		c_4_1 = max(c_4_1,a_4+b_1);
		c_4_2 = max(c_4_2,a_4+b_2);
		c_4_3 = max(c_4_3,a_4+b_3);
		c_4_4 = max(c_4_4,a_4+b_4);
		c_4_5 = max(c_4_5,a_4+b_5);
		c_4_6 = max(c_4_6,a_4+b_6);
		c_4_7 = max(c_4_7,a_4+b_7);
		c_4_8 = max(c_4_8,a_4+b_8);
		c_5_0 = max(c_5_0,a_5+b_0);
		c_5_1 = max(c_5_1,a_5+b_1);
		c_5_2 = max(c_5_2,a_5+b_2);
		c_5_3 = max(c_5_3,a_5+b_3);
		c_5_4 = max(c_5_4,a_5+b_4);
		c_5_5 = max(c_5_5,a_5+b_5);
		c_5_6 = max(c_5_6,a_5+b_6);
		c_5_7 = max(c_5_7,a_5+b_7);
		c_5_8 = max(c_5_8,a_5+b_8);
		c_6_0 = max(c_6_0,a_6+b_0);
		c_6_1 = max(c_6_1,a_6+b_1);
		c_6_2 = max(c_6_2,a_6+b_2);
		c_6_3 = max(c_6_3,a_6+b_3);
		c_6_4 = max(c_6_4,a_6+b_4);
		c_6_5 = max(c_6_5,a_6+b_5);
		c_6_6 = max(c_6_6,a_6+b_6);
		c_6_7 = max(c_6_7,a_6+b_7);
		c_6_8 = max(c_6_8,a_6+b_8);
		c_7_0 = max(c_7_0,a_7+b_0);
		c_7_1 = max(c_7_1,a_7+b_1);
		c_7_2 = max(c_7_2,a_7+b_2);
		c_7_3 = max(c_7_3,a_7+b_3);
		c_7_4 = max(c_7_4,a_7+b_4);
		c_7_5 = max(c_7_5,a_7+b_5);
		c_7_6 = max(c_7_6,a_7+b_6);
		c_7_7 = max(c_7_7,a_7+b_7);
		c_7_8 = max(c_7_8,a_7+b_8);
		c_8_0 = max(c_8_0,a_8+b_0);
		c_8_1 = max(c_8_1,a_8+b_1);
		c_8_2 = max(c_8_2,a_8+b_2);
		c_8_3 = max(c_8_3,a_8+b_3);
		c_8_4 = max(c_8_4,a_8+b_4);
		c_8_5 = max(c_8_5,a_8+b_5);
		c_8_6 = max(c_8_6,a_8+b_6);
		c_8_7 = max(c_8_7,a_8+b_7);
		c_8_8 = max(c_8_8,a_8+b_8);


		a_0 = max(a_0, a_0+10);
		a_1 = max(a_1, a_1+10);
		a_2 = max(a_2, a_2+10);
		a_3 = max(a_3, a_3+10);
		a_4 = max(a_4, a_4+10);
		a_5 = max(a_5, a_5+10);
		a_6 = max(a_6, a_6+10);
		a_7 = max(a_7, a_7+10);
		a_8 = max(a_8, a_8+10);


		b_0 = max(b_0, b_0+10);
		b_1 = max(b_1, b_1+10);
		b_2 = max(b_2, b_2+10);
		b_3 = max(b_3, b_3+10);
		b_4 = max(b_4, b_4+10);
		b_5 = max(b_5, b_5+10);
		b_6 = max(b_6, b_6+10);
		b_7 = max(b_7, b_7+10);
		b_8 = max(b_8, b_8+10);


	}


	C[(A_start_index+0)*C_width + B_start_index+0] = c_0_0;
	C[(A_start_index+0)*C_width + B_start_index+1] = c_0_1;
	C[(A_start_index+0)*C_width + B_start_index+2] = c_0_2;
	C[(A_start_index+0)*C_width + B_start_index+3] = c_0_3;
	C[(A_start_index+0)*C_width + B_start_index+4] = c_0_4;
	C[(A_start_index+0)*C_width + B_start_index+5] = c_0_5;
	C[(A_start_index+0)*C_width + B_start_index+6] = c_0_6;
	C[(A_start_index+0)*C_width + B_start_index+7] = c_0_7;
	C[(A_start_index+0)*C_width + B_start_index+8] = c_0_8;
	C[(A_start_index+1)*C_width + B_start_index+0] = c_1_0;
	C[(A_start_index+1)*C_width + B_start_index+1] = c_1_1;
	C[(A_start_index+1)*C_width + B_start_index+2] = c_1_2;
	C[(A_start_index+1)*C_width + B_start_index+3] = c_1_3;
	C[(A_start_index+1)*C_width + B_start_index+4] = c_1_4;
	C[(A_start_index+1)*C_width + B_start_index+5] = c_1_5;
	C[(A_start_index+1)*C_width + B_start_index+6] = c_1_6;
	C[(A_start_index+1)*C_width + B_start_index+7] = c_1_7;
	C[(A_start_index+1)*C_width + B_start_index+8] = c_1_8;
	C[(A_start_index+2)*C_width + B_start_index+0] = c_2_0;
	C[(A_start_index+2)*C_width + B_start_index+1] = c_2_1;
	C[(A_start_index+2)*C_width + B_start_index+2] = c_2_2;
	C[(A_start_index+2)*C_width + B_start_index+3] = c_2_3;
	C[(A_start_index+2)*C_width + B_start_index+4] = c_2_4;
	C[(A_start_index+2)*C_width + B_start_index+5] = c_2_5;
	C[(A_start_index+2)*C_width + B_start_index+6] = c_2_6;
	C[(A_start_index+2)*C_width + B_start_index+7] = c_2_7;
	C[(A_start_index+2)*C_width + B_start_index+8] = c_2_8;
	C[(A_start_index+3)*C_width + B_start_index+0] = c_3_0;
	C[(A_start_index+3)*C_width + B_start_index+1] = c_3_1;
	C[(A_start_index+3)*C_width + B_start_index+2] = c_3_2;
	C[(A_start_index+3)*C_width + B_start_index+3] = c_3_3;
	C[(A_start_index+3)*C_width + B_start_index+4] = c_3_4;
	C[(A_start_index+3)*C_width + B_start_index+5] = c_3_5;
	C[(A_start_index+3)*C_width + B_start_index+6] = c_3_6;
	C[(A_start_index+3)*C_width + B_start_index+7] = c_3_7;
	C[(A_start_index+3)*C_width + B_start_index+8] = c_3_8;
	C[(A_start_index+4)*C_width + B_start_index+0] = c_4_0;
	C[(A_start_index+4)*C_width + B_start_index+1] = c_4_1;
	C[(A_start_index+4)*C_width + B_start_index+2] = c_4_2;
	C[(A_start_index+4)*C_width + B_start_index+3] = c_4_3;
	C[(A_start_index+4)*C_width + B_start_index+4] = c_4_4;
	C[(A_start_index+4)*C_width + B_start_index+5] = c_4_5;
	C[(A_start_index+4)*C_width + B_start_index+6] = c_4_6;
	C[(A_start_index+4)*C_width + B_start_index+7] = c_4_7;
	C[(A_start_index+4)*C_width + B_start_index+8] = c_4_8;
	C[(A_start_index+5)*C_width + B_start_index+0] = c_5_0;
	C[(A_start_index+5)*C_width + B_start_index+1] = c_5_1;
	C[(A_start_index+5)*C_width + B_start_index+2] = c_5_2;
	C[(A_start_index+5)*C_width + B_start_index+3] = c_5_3;
	C[(A_start_index+5)*C_width + B_start_index+4] = c_5_4;
	C[(A_start_index+5)*C_width + B_start_index+5] = c_5_5;
	C[(A_start_index+5)*C_width + B_start_index+6] = c_5_6;
	C[(A_start_index+5)*C_width + B_start_index+7] = c_5_7;
	C[(A_start_index+5)*C_width + B_start_index+8] = c_5_8;
	C[(A_start_index+6)*C_width + B_start_index+0] = c_6_0;
	C[(A_start_index+6)*C_width + B_start_index+1] = c_6_1;
	C[(A_start_index+6)*C_width + B_start_index+2] = c_6_2;
	C[(A_start_index+6)*C_width + B_start_index+3] = c_6_3;
	C[(A_start_index+6)*C_width + B_start_index+4] = c_6_4;
	C[(A_start_index+6)*C_width + B_start_index+5] = c_6_5;
	C[(A_start_index+6)*C_width + B_start_index+6] = c_6_6;
	C[(A_start_index+6)*C_width + B_start_index+7] = c_6_7;
	C[(A_start_index+6)*C_width + B_start_index+8] = c_6_8;
	C[(A_start_index+7)*C_width + B_start_index+0] = c_7_0;
	C[(A_start_index+7)*C_width + B_start_index+1] = c_7_1;
	C[(A_start_index+7)*C_width + B_start_index+2] = c_7_2;
	C[(A_start_index+7)*C_width + B_start_index+3] = c_7_3;
	C[(A_start_index+7)*C_width + B_start_index+4] = c_7_4;
	C[(A_start_index+7)*C_width + B_start_index+5] = c_7_5;
	C[(A_start_index+7)*C_width + B_start_index+6] = c_7_6;
	C[(A_start_index+7)*C_width + B_start_index+7] = c_7_7;
	C[(A_start_index+7)*C_width + B_start_index+8] = c_7_8;
	C[(A_start_index+8)*C_width + B_start_index+0] = c_8_0;
	C[(A_start_index+8)*C_width + B_start_index+1] = c_8_1;
	C[(A_start_index+8)*C_width + B_start_index+2] = c_8_2;
	C[(A_start_index+8)*C_width + B_start_index+3] = c_8_3;
	C[(A_start_index+8)*C_width + B_start_index+4] = c_8_4;
	C[(A_start_index+8)*C_width + B_start_index+5] = c_8_5;
	C[(A_start_index+8)*C_width + B_start_index+6] = c_8_6;
	C[(A_start_index+8)*C_width + B_start_index+7] = c_8_7;
	C[(A_start_index+8)*C_width + B_start_index+8] = c_8_8;


}
