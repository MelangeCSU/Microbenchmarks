
#include <hip/hip_runtime.h>
/** size of A = 384
    size of B = 360
    gridDim = 60
    blockDim = 64
    k= 400000
    x = 6
**/

#define max(x,y)   ((x)>(y) ? (x) : (y))
__global__ void MultiplyVectors(const int* A, const int* B, int* C, int x, int k)
{
	int B_start_index = (blockIdx.x*gridDim.y + blockIdx.y)*x;
	int A_start_index = (threadIdx.x*blockDim.y + threadIdx.y)*x;
	int C_width = x*gridDim.x*gridDim.y;


	int t;
	int c_0_0, c_0_1, c_0_2, c_0_3, c_0_4, c_0_5, c_1_0, c_1_1, c_1_2, c_1_3, c_1_4, c_1_5, c_2_0, c_2_1, c_2_2, c_2_3, c_2_4, c_2_5, c_3_0, c_3_1, c_3_2, c_3_3, c_3_4, c_3_5, c_4_0, c_4_1, c_4_2, c_4_3, c_4_4, c_4_5, c_5_0, c_5_1, c_5_2, c_5_3, c_5_4, c_5_5;
	int a_0, a_1, a_2, a_3, a_4, a_5;
	int b_0, b_1, b_2, b_3, b_4, b_5;


	a_0 = A[A_start_index+0];
	a_1 = A[A_start_index+1];
	a_2 = A[A_start_index+2];
	a_3 = A[A_start_index+3];
	a_4 = A[A_start_index+4];
	a_5 = A[A_start_index+5];


	b_0 = B[B_start_index+0];
	b_1 = B[B_start_index+1];
	b_2 = B[B_start_index+2];
	b_3 = B[B_start_index+3];
	b_4 = B[B_start_index+4];
	b_5 = B[B_start_index+5];


	c_0_0 = 0;
	c_0_1 = 0;
	c_0_2 = 0;
	c_0_3 = 0;
	c_0_4 = 0;
	c_0_5 = 0;
	c_1_0 = 0;
	c_1_1 = 0;
	c_1_2 = 0;
	c_1_3 = 0;
	c_1_4 = 0;
	c_1_5 = 0;
	c_2_0 = 0;
	c_2_1 = 0;
	c_2_2 = 0;
	c_2_3 = 0;
	c_2_4 = 0;
	c_2_5 = 0;
	c_3_0 = 0;
	c_3_1 = 0;
	c_3_2 = 0;
	c_3_3 = 0;
	c_3_4 = 0;
	c_3_5 = 0;
	c_4_0 = 0;
	c_4_1 = 0;
	c_4_2 = 0;
	c_4_3 = 0;
	c_4_4 = 0;
	c_4_5 = 0;
	c_5_0 = 0;
	c_5_1 = 0;
	c_5_2 = 0;
	c_5_3 = 0;
	c_5_4 = 0;
	c_5_5 = 0;


	for (t = 0; t < 400000; t++) {
		c_0_0 = max(c_0_0 ,a_0+b_0);
		c_0_1 = max(c_0_1 ,a_0+b_1);
		c_0_2 = max(c_0_2 ,a_0+b_2);
		c_0_3 = max(c_0_3 ,a_0+b_3);
		c_0_4 = max(c_0_4 ,a_0+b_4);
		c_0_5 = max(c_0_5 ,a_0+b_5);
		c_1_0 = max(c_1_0 ,a_1+b_0);
		c_1_1 = max(c_1_1 ,a_1+b_1);
		c_1_2 = max(c_1_2 ,a_1+b_2);
		c_1_3 = max(c_1_3 ,a_1+b_3);
		c_1_4 = max(c_1_4 ,a_1+b_4);
		c_1_5 = max(c_1_5 ,a_1+b_5);
		c_2_0 = max(c_2_0 ,a_2+b_0);
		c_2_1 = max(c_2_1 ,a_2+b_1);
		c_2_2 = max(c_2_2 ,a_2+b_2);
		c_2_3 = max(c_2_3 ,a_2+b_3);
		c_2_4 = max(c_2_4 ,a_2+b_4);
		c_2_5 = max(c_2_5 ,a_2+b_5);
		c_3_0 = max(c_3_0 ,a_3+b_0);
		c_3_1 = max(c_3_1 ,a_3+b_1);
		c_3_2 = max(c_3_2 ,a_3+b_2);
		c_3_3 = max(c_3_3 ,a_3+b_3);
		c_3_4 = max(c_3_4 ,a_3+b_4);
		c_3_5 = max(c_3_5 ,a_3+b_5);
		c_4_0 = max(c_4_0 ,a_4+b_0);
		c_4_1 = max(c_4_1 ,a_4+b_1);
		c_4_2 = max(c_4_2 ,a_4+b_2);
		c_4_3 = max(c_4_3 ,a_4+b_3);
		c_4_4 = max(c_4_4 ,a_4+b_4);
		c_4_5 = max(c_4_5 ,a_4+b_5);
		c_5_0 = max(c_5_0 ,a_5+b_0);
		c_5_1 = max(c_5_1 ,a_5+b_1);
		c_5_2 = max(c_5_2 ,a_5+b_2);
		c_5_3 = max(c_5_3 ,a_5+b_3);
		c_5_4 = max(c_5_4 ,a_5+b_4);
		c_5_5 = max(c_5_5 ,a_5+b_5);


		a_0 = max(a_0,a_0+b_5);
		a_1 = max(a_1,a_1+b_4);
		a_2 = max(a_2,a_2+b_3);
		a_3 = max(a_3,a_3+b_2);
		a_4 = max(a_4,a_4+b_1);
		a_5 = max(a_5,a_5+b_0);


		b_0 =  max(b_0,b_0+a_5);
		b_1 =  max(b_1,b_1+a_4);
		b_2 =  max(b_2,b_2+a_3);
		b_3 =  max(b_3,b_3+a_2);
		b_4 =  max(b_4,b_4+a_1);
		b_5 =  max(b_5,b_5+a_0);


	}


	C[(A_start_index+0)*C_width + B_start_index+0] = c_0_0;
	C[(A_start_index+0)*C_width + B_start_index+1] = c_0_1;
	C[(A_start_index+0)*C_width + B_start_index+2] = c_0_2;
	C[(A_start_index+0)*C_width + B_start_index+3] = c_0_3;
	C[(A_start_index+0)*C_width + B_start_index+4] = c_0_4;
	C[(A_start_index+0)*C_width + B_start_index+5] = c_0_5;
	C[(A_start_index+1)*C_width + B_start_index+0] = c_1_0;
	C[(A_start_index+1)*C_width + B_start_index+1] = c_1_1;
	C[(A_start_index+1)*C_width + B_start_index+2] = c_1_2;
	C[(A_start_index+1)*C_width + B_start_index+3] = c_1_3;
	C[(A_start_index+1)*C_width + B_start_index+4] = c_1_4;
	C[(A_start_index+1)*C_width + B_start_index+5] = c_1_5;
	C[(A_start_index+2)*C_width + B_start_index+0] = c_2_0;
	C[(A_start_index+2)*C_width + B_start_index+1] = c_2_1;
	C[(A_start_index+2)*C_width + B_start_index+2] = c_2_2;
	C[(A_start_index+2)*C_width + B_start_index+3] = c_2_3;
	C[(A_start_index+2)*C_width + B_start_index+4] = c_2_4;
	C[(A_start_index+2)*C_width + B_start_index+5] = c_2_5;
	C[(A_start_index+3)*C_width + B_start_index+0] = c_3_0;
	C[(A_start_index+3)*C_width + B_start_index+1] = c_3_1;
	C[(A_start_index+3)*C_width + B_start_index+2] = c_3_2;
	C[(A_start_index+3)*C_width + B_start_index+3] = c_3_3;
	C[(A_start_index+3)*C_width + B_start_index+4] = c_3_4;
	C[(A_start_index+3)*C_width + B_start_index+5] = c_3_5;
	C[(A_start_index+4)*C_width + B_start_index+0] = c_4_0;
	C[(A_start_index+4)*C_width + B_start_index+1] = c_4_1;
	C[(A_start_index+4)*C_width + B_start_index+2] = c_4_2;
	C[(A_start_index+4)*C_width + B_start_index+3] = c_4_3;
	C[(A_start_index+4)*C_width + B_start_index+4] = c_4_4;
	C[(A_start_index+4)*C_width + B_start_index+5] = c_4_5;
	C[(A_start_index+5)*C_width + B_start_index+0] = c_5_0;
	C[(A_start_index+5)*C_width + B_start_index+1] = c_5_1;
	C[(A_start_index+5)*C_width + B_start_index+2] = c_5_2;
	C[(A_start_index+5)*C_width + B_start_index+3] = c_5_3;
	C[(A_start_index+5)*C_width + B_start_index+4] = c_5_4;
	C[(A_start_index+5)*C_width + B_start_index+5] = c_5_5;


}
