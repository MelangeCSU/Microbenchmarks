#include "hip/hip_runtime.h"

#include <stdio.h>
#include <cutil.h>
#include "vecmultKernel.h"

// Defines
//#define GridWidth 60
//#define BlockWidth 128

// Variables for host and device vectors.
int* h_A; 
int* h_B; 
int* h_C; 
int* d_A; 
int* d_B; 
int* d_C; 

// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{
    int ValuesPerThread; // number of values per thread
    int N; //Vector size
	int k; // no. of repeatitions
	int gridWidth = 60;
	int blockWidth = 1;

	// Parse arguments.
    if(argc != 5){
     printf("Usage: %s ValuesPerThread Iterations\n", argv[0]);
     printf("ValuesPerThread is the number of values added by each thread.\n");
     printf("Total vector size is 128 * 60 * this value.\n");
     printf("Iterations is the number of repeatitions done by each thread.\n");
     exit(0);
    } else {
      sscanf(argv[1], "%d", &ValuesPerThread);
      sscanf(argv[2], "%d", &k);
      sscanf(argv[3], "%d", &gridWidth);
      sscanf(argv[4], "%d", &blockWidth);
    }      

	int size_A = blockWidth * ValuesPerThread;
	int size_B = gridWidth * ValuesPerThread;

	printf("Size of A: %d, Size of B: %d\n", size_A, size_B);
	//printf("1st bf\n");
    // Determine the number of threads .
    // N is the total number of values to be in a vector
//    N = ValuesPerThread * gridWidth * blockWidth;
//    printf("Total vector size: %d : Iterations: %d\n", N, k); 
    // size_t is the total number of bytes for a vector.
//    size_t size = N * sizeof(float);

    // Tell CUDA how big to make the grid and thread blocks.
    // Since this is a vector addition problem,
    // grid and thread block are both one-dimensional.
    dim3 dimGrid(gridWidth);                    
    dim3 dimBlock(blockWidth);                 

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size_A*sizeof(int));
    if (h_A == 0) Cleanup(false);
    h_B = (int*)malloc(size_B*sizeof(int));
    if (h_B == 0) Cleanup(false);
    h_C = (int*)malloc(size_A*sizeof(int)*size_B*sizeof(int));
    if (h_C == 0) Cleanup(false);
	
//	printf("1 \n");
	//printf("2nd  bp\n");
    // Allocate vectors in device memory.
    hipError_t error;
    error = hipMalloc((void**)&d_A, size_A*sizeof(int));
    if (error != hipSuccess) Cleanup(false);
	//printf("1.1\n");
    error = hipMalloc((void**)&d_B, size_B*sizeof(int));
    if (error != hipSuccess) Cleanup(false);
//	printf("1.2\n");
    error = hipMalloc((void**)&d_C, size_A*sizeof(int)*size_B*sizeof(int));
    if (error != hipSuccess) Cleanup(false);

//	printf("2\n");
    // Initialize host vectors h_A and h_B
    int i, j;
    for(i=0; i <size_A; ++i){
     h_A[i] = (int)i;
    }
    for(i=0; i <size_B; ++i){
     h_B[i] = (int)(N-i);   
    }
	
	//printf("3rd bp\n");
    // Copy host vectors h_A and h_B to device vectores d_A and d_B



    error = hipMemcpy(d_A, h_A, size_A*sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);
    error = hipMemcpy(d_B, h_B, size_B*sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);
	

    MultiplyVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);
    error = hipGetLastError();
//printf("4th bp\n");
    if (error != hipSuccess) {
		printf("W: %s\n", hipGetErrorString(error));
		Cleanup(false);
	}
    hipDeviceSynchronize();


//	printf("4\n");
    // Initialize timer

 // printf("Starting Kernel\n");
    unsigned int timer = 0;
    cutCreateTimer(&timer);
    cutStartTimer(timer);
  
    // Invoke kernel
    MultiplyVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
		Cleanup(false);
	}

//	printf("5\n");
	// Compute elapsed time 
    hipDeviceSynchronize();
    cutStopTimer(timer);
    float time = cutGetTimerValue(timer);

	// Compute floating point operations per second.
    //double nFlops = size_A*size_B*k*2 + 4*k*6*60;
    double nFlops = (double)size_A*(double)size_B*(double)k*(double)2 + (double)4*(double)k*(double)ValuesPerThread*(double)gridWidth*(double)blockWidth;
    //long nFlops = (int)size_A*(int)size_B*(int)k*(int)2;
	//printf("nFlops=%ld\n",nFlops);
	//nFlops+=(int)4*ValuesPerThread*(int)k*gridWidth*blockWidth;
	//printf("nFlops=%ld\n",nFlops);
    float nFlopsPerSec = 1e3*float(nFlops)/time;
    float nGFlopsPerSec = nFlopsPerSec*1e-9;
	//printf("%f :: %f", (double)size_A*(double)size_B*(double)k*(double)2, (double)2*(double)k*(double)ValuesPerThread*(double)gridWidth*(double)blockWidth);
	// Compute transfer rates.
    int nBytes = size_A*sizeof(int) + size_B*sizeof(int) + size_A*sizeof(int)*size_B*sizeof(int); // 2N words in, N*N word out
    float nBytesPerSec = 1e3*(float)nBytes/time;
    float nGBytesPerSec = nBytesPerSec*1e-9;

	// Report timing data.
    printf( "Time: %f (ms), GFLOPS: %f, GBytesS: %f\n", 
             time, nGFlopsPerSec, nGBytesPerSec);
     
    // Copy result from device memory to host memory
    error = hipMemcpy(h_C, d_C, size_A*sizeof(int)*size_B*sizeof(int), hipMemcpyDeviceToHost);
    if (error != hipSuccess) Cleanup(false);

    // Verify & report result
    for (i = 0; i < size_A; ++i) {
    	for (j = 0; j < size_B; ++j) {
			int val = h_C[i*size_B+j];
			if (fabs(val - h_A[i]*h_B[j]) > 1e-5) {
				printf("Result error: i=%d, j=%d, expected %d, got %d\n", i, j, h_A[i]*h_B[j], val);
				break;
			}
		}
		if (j != size_B) {
			break;
		}
    }
    printf("Test %s \n", (i == size_A && j == size_B) ? "PASSED" : "FAILED");

	// Clean up and exit.
    cutDeleteTimer( timer);
    Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    error = hipDeviceReset();
   checkCUDAError("tmp"); 
    if (!noError || error != hipSuccess)
        printf("error: %s cuda malloc or cuda thread exit failed \n", hipGetErrorString(hipGetLastError()));
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
		printf("Error");
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      exit(-1);
    }                         
}


