
#include <hip/hip_runtime.h>
/*Author: Swetha Varadarajan 6/10/2015
 Adapted from Waruna Ranasinghe's summer 2014 work on kpdp micro-benchmarking 
 Code generated from intmazaddF2CG.c 
 Parameters list
size of A = 2880
    size of B = 1650
    gridDim = 165
    blockDim = 288
    Iterations k= 800000
   Values per thread x = 10
*/

__global__ void IntmaxaddF2(const int* A, const int* B, int* C, int x, int k)
{
	int B_start_index = (blockIdx.x*gridDim.y + blockIdx.y)*x;
	int A_start_index = (threadIdx.x*blockDim.y + threadIdx.y)*x;
	int C_width = x*gridDim.x*gridDim.y;


	int i,j,t;
	__shared__ int c[288][10][10];
	__shared__ int a[288][10];
	__shared__ int b[10];


	a[threadIdx.x][0] = A[A_start_index+0];
	a[threadIdx.x][1] = A[A_start_index+1];
	a[threadIdx.x][2] = A[A_start_index+2];
	a[threadIdx.x][3] = A[A_start_index+3];
	a[threadIdx.x][4] = A[A_start_index+4];
	a[threadIdx.x][5] = A[A_start_index+5];
	a[threadIdx.x][6] = A[A_start_index+6];
	a[threadIdx.x][7] = A[A_start_index+7];
	a[threadIdx.x][8] = A[A_start_index+8];
	a[threadIdx.x][9] = A[A_start_index+9];


	b[0] = B[B_start_index+0];
	b[1] = B[B_start_index+1];
	b[2] = B[B_start_index+2];
	b[3] = B[B_start_index+3];
	b[4] = B[B_start_index+4];
	b[5] = B[B_start_index+5];
	b[6] = B[B_start_index+6];
	b[7] = B[B_start_index+7];
	b[8] = B[B_start_index+8];
	b[9] = B[B_start_index+9];


	#pragma unroll
	for(i=0;i<x;i++)
	{
		for(j=0;j<x;j++)
		{
			c[threadIdx.x][i][j]=0;
		}
	}


	for (t = 0; t < k; t++) {
	#pragma unroll
		for(i=0;i<x;i++)
			{
				for(j=0;j<x;j++)
			{
				c[threadIdx.x][i][j] = max(a[threadIdx.x][j],(b[j]+c[threadIdx.x][i][j]));
			}
		}


		a[threadIdx.x][0] = max(a[threadIdx.x][0],(b[0]+c[threadIdx.x][0][0]));
		a[threadIdx.x][1] = max(a[threadIdx.x][1],(b[1]+c[threadIdx.x][0][1]));
		a[threadIdx.x][2] = max(a[threadIdx.x][2],(b[2]+c[threadIdx.x][0][2]));
		a[threadIdx.x][3] = max(a[threadIdx.x][3],(b[3]+c[threadIdx.x][0][3]));
		a[threadIdx.x][4] = max(a[threadIdx.x][4],(b[4]+c[threadIdx.x][0][4]));
		a[threadIdx.x][5] = max(a[threadIdx.x][5],(b[5]+c[threadIdx.x][0][5]));
		a[threadIdx.x][6] = max(a[threadIdx.x][6],(b[6]+c[threadIdx.x][0][6]));
		a[threadIdx.x][7] = max(a[threadIdx.x][7],(b[7]+c[threadIdx.x][0][7]));
		a[threadIdx.x][8] = max(a[threadIdx.x][8],(b[8]+c[threadIdx.x][0][8]));
		a[threadIdx.x][9] = max(a[threadIdx.x][9],(b[9]+c[threadIdx.x][0][9]));


		b[0] = max(b[0],(a[threadIdx.x][9]+c[threadIdx.x][1][0]));
		b[1] = max(b[1],(a[threadIdx.x][8]+c[threadIdx.x][1][1]));
		b[2] = max(b[2],(a[threadIdx.x][7]+c[threadIdx.x][1][2]));
		b[3] = max(b[3],(a[threadIdx.x][6]+c[threadIdx.x][1][3]));
		b[4] = max(b[4],(a[threadIdx.x][5]+c[threadIdx.x][1][4]));
		b[5] = max(b[5],(a[threadIdx.x][4]+c[threadIdx.x][1][5]));
		b[6] = max(b[6],(a[threadIdx.x][3]+c[threadIdx.x][1][6]));
		b[7] = max(b[7],(a[threadIdx.x][2]+c[threadIdx.x][1][7]));
		b[8] = max(b[8],(a[threadIdx.x][1]+c[threadIdx.x][1][8]));
		b[9] = max(b[9],(a[threadIdx.x][0]+c[threadIdx.x][1][9]));




	}


	C[(A_start_index+0)*C_width + B_start_index+0] = c[threadIdx.x][0][0];
	C[(A_start_index+0)*C_width + B_start_index+1] = c[threadIdx.x][0][1];
	C[(A_start_index+0)*C_width + B_start_index+2] = c[threadIdx.x][0][2];
	C[(A_start_index+0)*C_width + B_start_index+3] = c[threadIdx.x][0][3];
	C[(A_start_index+0)*C_width + B_start_index+4] = c[threadIdx.x][0][4];
	C[(A_start_index+0)*C_width + B_start_index+5] = c[threadIdx.x][0][5];
	C[(A_start_index+0)*C_width + B_start_index+6] = c[threadIdx.x][0][6];
	C[(A_start_index+0)*C_width + B_start_index+7] = c[threadIdx.x][0][7];
	C[(A_start_index+0)*C_width + B_start_index+8] = c[threadIdx.x][0][8];
	C[(A_start_index+0)*C_width + B_start_index+9] = c[threadIdx.x][0][9];
	C[(A_start_index+1)*C_width + B_start_index+0] = c[threadIdx.x][1][0];
	C[(A_start_index+1)*C_width + B_start_index+1] = c[threadIdx.x][1][1];
	C[(A_start_index+1)*C_width + B_start_index+2] = c[threadIdx.x][1][2];
	C[(A_start_index+1)*C_width + B_start_index+3] = c[threadIdx.x][1][3];
	C[(A_start_index+1)*C_width + B_start_index+4] = c[threadIdx.x][1][4];
	C[(A_start_index+1)*C_width + B_start_index+5] = c[threadIdx.x][1][5];
	C[(A_start_index+1)*C_width + B_start_index+6] = c[threadIdx.x][1][6];
	C[(A_start_index+1)*C_width + B_start_index+7] = c[threadIdx.x][1][7];
	C[(A_start_index+1)*C_width + B_start_index+8] = c[threadIdx.x][1][8];
	C[(A_start_index+1)*C_width + B_start_index+9] = c[threadIdx.x][1][9];
	C[(A_start_index+2)*C_width + B_start_index+0] = c[threadIdx.x][2][0];
	C[(A_start_index+2)*C_width + B_start_index+1] = c[threadIdx.x][2][1];
	C[(A_start_index+2)*C_width + B_start_index+2] = c[threadIdx.x][2][2];
	C[(A_start_index+2)*C_width + B_start_index+3] = c[threadIdx.x][2][3];
	C[(A_start_index+2)*C_width + B_start_index+4] = c[threadIdx.x][2][4];
	C[(A_start_index+2)*C_width + B_start_index+5] = c[threadIdx.x][2][5];
	C[(A_start_index+2)*C_width + B_start_index+6] = c[threadIdx.x][2][6];
	C[(A_start_index+2)*C_width + B_start_index+7] = c[threadIdx.x][2][7];
	C[(A_start_index+2)*C_width + B_start_index+8] = c[threadIdx.x][2][8];
	C[(A_start_index+2)*C_width + B_start_index+9] = c[threadIdx.x][2][9];
	C[(A_start_index+3)*C_width + B_start_index+0] = c[threadIdx.x][3][0];
	C[(A_start_index+3)*C_width + B_start_index+1] = c[threadIdx.x][3][1];
	C[(A_start_index+3)*C_width + B_start_index+2] = c[threadIdx.x][3][2];
	C[(A_start_index+3)*C_width + B_start_index+3] = c[threadIdx.x][3][3];
	C[(A_start_index+3)*C_width + B_start_index+4] = c[threadIdx.x][3][4];
	C[(A_start_index+3)*C_width + B_start_index+5] = c[threadIdx.x][3][5];
	C[(A_start_index+3)*C_width + B_start_index+6] = c[threadIdx.x][3][6];
	C[(A_start_index+3)*C_width + B_start_index+7] = c[threadIdx.x][3][7];
	C[(A_start_index+3)*C_width + B_start_index+8] = c[threadIdx.x][3][8];
	C[(A_start_index+3)*C_width + B_start_index+9] = c[threadIdx.x][3][9];
	C[(A_start_index+4)*C_width + B_start_index+0] = c[threadIdx.x][4][0];
	C[(A_start_index+4)*C_width + B_start_index+1] = c[threadIdx.x][4][1];
	C[(A_start_index+4)*C_width + B_start_index+2] = c[threadIdx.x][4][2];
	C[(A_start_index+4)*C_width + B_start_index+3] = c[threadIdx.x][4][3];
	C[(A_start_index+4)*C_width + B_start_index+4] = c[threadIdx.x][4][4];
	C[(A_start_index+4)*C_width + B_start_index+5] = c[threadIdx.x][4][5];
	C[(A_start_index+4)*C_width + B_start_index+6] = c[threadIdx.x][4][6];
	C[(A_start_index+4)*C_width + B_start_index+7] = c[threadIdx.x][4][7];
	C[(A_start_index+4)*C_width + B_start_index+8] = c[threadIdx.x][4][8];
	C[(A_start_index+4)*C_width + B_start_index+9] = c[threadIdx.x][4][9];
	C[(A_start_index+5)*C_width + B_start_index+0] = c[threadIdx.x][5][0];
	C[(A_start_index+5)*C_width + B_start_index+1] = c[threadIdx.x][5][1];
	C[(A_start_index+5)*C_width + B_start_index+2] = c[threadIdx.x][5][2];
	C[(A_start_index+5)*C_width + B_start_index+3] = c[threadIdx.x][5][3];
	C[(A_start_index+5)*C_width + B_start_index+4] = c[threadIdx.x][5][4];
	C[(A_start_index+5)*C_width + B_start_index+5] = c[threadIdx.x][5][5];
	C[(A_start_index+5)*C_width + B_start_index+6] = c[threadIdx.x][5][6];
	C[(A_start_index+5)*C_width + B_start_index+7] = c[threadIdx.x][5][7];
	C[(A_start_index+5)*C_width + B_start_index+8] = c[threadIdx.x][5][8];
	C[(A_start_index+5)*C_width + B_start_index+9] = c[threadIdx.x][5][9];
	C[(A_start_index+6)*C_width + B_start_index+0] = c[threadIdx.x][6][0];
	C[(A_start_index+6)*C_width + B_start_index+1] = c[threadIdx.x][6][1];
	C[(A_start_index+6)*C_width + B_start_index+2] = c[threadIdx.x][6][2];
	C[(A_start_index+6)*C_width + B_start_index+3] = c[threadIdx.x][6][3];
	C[(A_start_index+6)*C_width + B_start_index+4] = c[threadIdx.x][6][4];
	C[(A_start_index+6)*C_width + B_start_index+5] = c[threadIdx.x][6][5];
	C[(A_start_index+6)*C_width + B_start_index+6] = c[threadIdx.x][6][6];
	C[(A_start_index+6)*C_width + B_start_index+7] = c[threadIdx.x][6][7];
	C[(A_start_index+6)*C_width + B_start_index+8] = c[threadIdx.x][6][8];
	C[(A_start_index+6)*C_width + B_start_index+9] = c[threadIdx.x][6][9];
	C[(A_start_index+7)*C_width + B_start_index+0] = c[threadIdx.x][7][0];
	C[(A_start_index+7)*C_width + B_start_index+1] = c[threadIdx.x][7][1];
	C[(A_start_index+7)*C_width + B_start_index+2] = c[threadIdx.x][7][2];
	C[(A_start_index+7)*C_width + B_start_index+3] = c[threadIdx.x][7][3];
	C[(A_start_index+7)*C_width + B_start_index+4] = c[threadIdx.x][7][4];
	C[(A_start_index+7)*C_width + B_start_index+5] = c[threadIdx.x][7][5];
	C[(A_start_index+7)*C_width + B_start_index+6] = c[threadIdx.x][7][6];
	C[(A_start_index+7)*C_width + B_start_index+7] = c[threadIdx.x][7][7];
	C[(A_start_index+7)*C_width + B_start_index+8] = c[threadIdx.x][7][8];
	C[(A_start_index+7)*C_width + B_start_index+9] = c[threadIdx.x][7][9];
	C[(A_start_index+8)*C_width + B_start_index+0] = c[threadIdx.x][8][0];
	C[(A_start_index+8)*C_width + B_start_index+1] = c[threadIdx.x][8][1];
	C[(A_start_index+8)*C_width + B_start_index+2] = c[threadIdx.x][8][2];
	C[(A_start_index+8)*C_width + B_start_index+3] = c[threadIdx.x][8][3];
	C[(A_start_index+8)*C_width + B_start_index+4] = c[threadIdx.x][8][4];
	C[(A_start_index+8)*C_width + B_start_index+5] = c[threadIdx.x][8][5];
	C[(A_start_index+8)*C_width + B_start_index+6] = c[threadIdx.x][8][6];
	C[(A_start_index+8)*C_width + B_start_index+7] = c[threadIdx.x][8][7];
	C[(A_start_index+8)*C_width + B_start_index+8] = c[threadIdx.x][8][8];
	C[(A_start_index+8)*C_width + B_start_index+9] = c[threadIdx.x][8][9];
	C[(A_start_index+9)*C_width + B_start_index+0] = c[threadIdx.x][9][0];
	C[(A_start_index+9)*C_width + B_start_index+1] = c[threadIdx.x][9][1];
	C[(A_start_index+9)*C_width + B_start_index+2] = c[threadIdx.x][9][2];
	C[(A_start_index+9)*C_width + B_start_index+3] = c[threadIdx.x][9][3];
	C[(A_start_index+9)*C_width + B_start_index+4] = c[threadIdx.x][9][4];
	C[(A_start_index+9)*C_width + B_start_index+5] = c[threadIdx.x][9][5];
	C[(A_start_index+9)*C_width + B_start_index+6] = c[threadIdx.x][9][6];
	C[(A_start_index+9)*C_width + B_start_index+7] = c[threadIdx.x][9][7];
	C[(A_start_index+9)*C_width + B_start_index+8] = c[threadIdx.x][9][8];
	C[(A_start_index+9)*C_width + B_start_index+9] = c[threadIdx.x][9][9];


}
