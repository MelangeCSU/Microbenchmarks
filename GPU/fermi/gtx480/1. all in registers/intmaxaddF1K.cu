
#include <hip/hip_runtime.h>
/*Author: Swetha Varadarajan 6/10/2015
 Adapted from Waruna Ranasinghe's summer 2014 work on kpdp micro-benchmarking 
 Code generated from intmazaddF1CG.c 
 Parameters list
size of A = 2048
    size of B = 3720
    gridDim = 465
    blockDim = 256
    Iterations k= 800000
   Values per thread x = 8
*/

__global__ void IntmaxaddF1(const int* A, const int* B, int* C, int x, int k)
{
	int B_start_index = (blockIdx.x*gridDim.y + blockIdx.y)*x;
	int A_start_index = (threadIdx.x*blockDim.y + threadIdx.y)*x;
	int C_width = x*gridDim.x*gridDim.y;


	int t;
	int c_0_0, c_0_1, c_0_2, c_0_3, c_0_4, c_0_5, c_0_6, c_0_7, c_1_0, c_1_1, c_1_2, c_1_3, c_1_4, c_1_5, c_1_6, c_1_7, c_2_0, c_2_1, c_2_2, c_2_3, c_2_4, c_2_5, c_2_6, c_2_7, c_3_0, c_3_1, c_3_2, c_3_3, c_3_4, c_3_5, c_3_6, c_3_7, c_4_0, c_4_1, c_4_2, c_4_3, c_4_4, c_4_5, c_4_6, c_4_7, c_5_0, c_5_1, c_5_2, c_5_3, c_5_4, c_5_5, c_5_6, c_5_7, c_6_0, c_6_1, c_6_2, c_6_3, c_6_4, c_6_5, c_6_6, c_6_7, c_7_0, c_7_1, c_7_2, c_7_3, c_7_4, c_7_5, c_7_6, c_7_7;
	int a_0, a_1, a_2, a_3, a_4, a_5, a_6, a_7;
	int b_0, b_1, b_2, b_3, b_4, b_5, b_6, b_7;


	a_0 = A[A_start_index+0];
	a_1 = A[A_start_index+1];
	a_2 = A[A_start_index+2];
	a_3 = A[A_start_index+3];
	a_4 = A[A_start_index+4];
	a_5 = A[A_start_index+5];
	a_6 = A[A_start_index+6];
	a_7 = A[A_start_index+7];


	b_0 = B[B_start_index+0];
	b_1 = B[B_start_index+1];
	b_2 = B[B_start_index+2];
	b_3 = B[B_start_index+3];
	b_4 = B[B_start_index+4];
	b_5 = B[B_start_index+5];
	b_6 = B[B_start_index+6];
	b_7 = B[B_start_index+7];


	c_0_0 = 0;
	c_0_1 = 0;
	c_0_2 = 0;
	c_0_3 = 0;
	c_0_4 = 0;
	c_0_5 = 0;
	c_0_6 = 0;
	c_0_7 = 0;
	c_1_0 = 0;
	c_1_1 = 0;
	c_1_2 = 0;
	c_1_3 = 0;
	c_1_4 = 0;
	c_1_5 = 0;
	c_1_6 = 0;
	c_1_7 = 0;
	c_2_0 = 0;
	c_2_1 = 0;
	c_2_2 = 0;
	c_2_3 = 0;
	c_2_4 = 0;
	c_2_5 = 0;
	c_2_6 = 0;
	c_2_7 = 0;
	c_3_0 = 0;
	c_3_1 = 0;
	c_3_2 = 0;
	c_3_3 = 0;
	c_3_4 = 0;
	c_3_5 = 0;
	c_3_6 = 0;
	c_3_7 = 0;
	c_4_0 = 0;
	c_4_1 = 0;
	c_4_2 = 0;
	c_4_3 = 0;
	c_4_4 = 0;
	c_4_5 = 0;
	c_4_6 = 0;
	c_4_7 = 0;
	c_5_0 = 0;
	c_5_1 = 0;
	c_5_2 = 0;
	c_5_3 = 0;
	c_5_4 = 0;
	c_5_5 = 0;
	c_5_6 = 0;
	c_5_7 = 0;
	c_6_0 = 0;
	c_6_1 = 0;
	c_6_2 = 0;
	c_6_3 = 0;
	c_6_4 = 0;
	c_6_5 = 0;
	c_6_6 = 0;
	c_6_7 = 0;
	c_7_0 = 0;
	c_7_1 = 0;
	c_7_2 = 0;
	c_7_3 = 0;
	c_7_4 = 0;
	c_7_5 = 0;
	c_7_6 = 0;
	c_7_7 = 0;


	for (t = 0; t < k; t++) {
		c_0_0 = max(a_0,c_0_0+b_0);
		c_0_1 = max(a_0,c_0_1+b_1);
		c_0_2 = max(a_0,c_0_2+b_2);
		c_0_3 = max(a_0,c_0_3+b_3);
		c_0_4 = max(a_0,c_0_4+b_4);
		c_0_5 = max(a_0,c_0_5+b_5);
		c_0_6 = max(a_0,c_0_6+b_6);
		c_0_7 = max(a_0,c_0_7+b_7);
		c_1_0 = max(a_1,c_1_0+b_0);
		c_1_1 = max(a_1,c_1_1+b_1);
		c_1_2 = max(a_1,c_1_2+b_2);
		c_1_3 = max(a_1,c_1_3+b_3);
		c_1_4 = max(a_1,c_1_4+b_4);
		c_1_5 = max(a_1,c_1_5+b_5);
		c_1_6 = max(a_1,c_1_6+b_6);
		c_1_7 = max(a_1,c_1_7+b_7);
		c_2_0 = max(a_2,c_2_0+b_0);
		c_2_1 = max(a_2,c_2_1+b_1);
		c_2_2 = max(a_2,c_2_2+b_2);
		c_2_3 = max(a_2,c_2_3+b_3);
		c_2_4 = max(a_2,c_2_4+b_4);
		c_2_5 = max(a_2,c_2_5+b_5);
		c_2_6 = max(a_2,c_2_6+b_6);
		c_2_7 = max(a_2,c_2_7+b_7);
		c_3_0 = max(a_3,c_3_0+b_0);
		c_3_1 = max(a_3,c_3_1+b_1);
		c_3_2 = max(a_3,c_3_2+b_2);
		c_3_3 = max(a_3,c_3_3+b_3);
		c_3_4 = max(a_3,c_3_4+b_4);
		c_3_5 = max(a_3,c_3_5+b_5);
		c_3_6 = max(a_3,c_3_6+b_6);
		c_3_7 = max(a_3,c_3_7+b_7);
		c_4_0 = max(a_4,c_4_0+b_0);
		c_4_1 = max(a_4,c_4_1+b_1);
		c_4_2 = max(a_4,c_4_2+b_2);
		c_4_3 = max(a_4,c_4_3+b_3);
		c_4_4 = max(a_4,c_4_4+b_4);
		c_4_5 = max(a_4,c_4_5+b_5);
		c_4_6 = max(a_4,c_4_6+b_6);
		c_4_7 = max(a_4,c_4_7+b_7);
		c_5_0 = max(a_5,c_5_0+b_0);
		c_5_1 = max(a_5,c_5_1+b_1);
		c_5_2 = max(a_5,c_5_2+b_2);
		c_5_3 = max(a_5,c_5_3+b_3);
		c_5_4 = max(a_5,c_5_4+b_4);
		c_5_5 = max(a_5,c_5_5+b_5);
		c_5_6 = max(a_5,c_5_6+b_6);
		c_5_7 = max(a_5,c_5_7+b_7);
		c_6_0 = max(a_6,c_6_0+b_0);
		c_6_1 = max(a_6,c_6_1+b_1);
		c_6_2 = max(a_6,c_6_2+b_2);
		c_6_3 = max(a_6,c_6_3+b_3);
		c_6_4 = max(a_6,c_6_4+b_4);
		c_6_5 = max(a_6,c_6_5+b_5);
		c_6_6 = max(a_6,c_6_6+b_6);
		c_6_7 = max(a_6,c_6_7+b_7);
		c_7_0 = max(a_7,c_7_0+b_0);
		c_7_1 = max(a_7,c_7_1+b_1);
		c_7_2 = max(a_7,c_7_2+b_2);
		c_7_3 = max(a_7,c_7_3+b_3);
		c_7_4 = max(a_7,c_7_4+b_4);
		c_7_5 = max(a_7,c_7_5+b_5);
		c_7_6 = max(a_7,c_7_6+b_6);
		c_7_7 = max(a_7,c_7_7+b_7);


		a_0 = max(a_0,b_7+c_0_7);
		a_1 = max(a_1,b_6+c_1_6);
		a_2 = max(a_2,b_5+c_2_5);
		a_3 = max(a_3,b_4+c_3_4);
		a_4 = max(a_4,b_3+c_4_3);
		a_5 = max(a_5,b_2+c_5_2);
		a_6 = max(a_6,b_1+c_6_1);
		a_7 = max(a_7,b_0+c_7_0);


		b_0 =  max(b_0,a_7+c_0_7);
		b_1 =  max(b_1,a_6+c_1_6);
		b_2 =  max(b_2,a_5+c_2_5);
		b_3 =  max(b_3,a_4+c_3_4);
		b_4 =  max(b_4,a_3+c_4_3);
		b_5 =  max(b_5,a_2+c_5_2);
		b_6 =  max(b_6,a_1+c_6_1);
		b_7 =  max(b_7,a_0+c_7_0);


	}


	C[(A_start_index+0)*C_width + B_start_index+0] = c_0_0;
	C[(A_start_index+0)*C_width + B_start_index+1] = c_0_1;
	C[(A_start_index+0)*C_width + B_start_index+2] = c_0_2;
	C[(A_start_index+0)*C_width + B_start_index+3] = c_0_3;
	C[(A_start_index+0)*C_width + B_start_index+4] = c_0_4;
	C[(A_start_index+0)*C_width + B_start_index+5] = c_0_5;
	C[(A_start_index+0)*C_width + B_start_index+6] = c_0_6;
	C[(A_start_index+0)*C_width + B_start_index+7] = c_0_7;
	C[(A_start_index+1)*C_width + B_start_index+0] = c_1_0;
	C[(A_start_index+1)*C_width + B_start_index+1] = c_1_1;
	C[(A_start_index+1)*C_width + B_start_index+2] = c_1_2;
	C[(A_start_index+1)*C_width + B_start_index+3] = c_1_3;
	C[(A_start_index+1)*C_width + B_start_index+4] = c_1_4;
	C[(A_start_index+1)*C_width + B_start_index+5] = c_1_5;
	C[(A_start_index+1)*C_width + B_start_index+6] = c_1_6;
	C[(A_start_index+1)*C_width + B_start_index+7] = c_1_7;
	C[(A_start_index+2)*C_width + B_start_index+0] = c_2_0;
	C[(A_start_index+2)*C_width + B_start_index+1] = c_2_1;
	C[(A_start_index+2)*C_width + B_start_index+2] = c_2_2;
	C[(A_start_index+2)*C_width + B_start_index+3] = c_2_3;
	C[(A_start_index+2)*C_width + B_start_index+4] = c_2_4;
	C[(A_start_index+2)*C_width + B_start_index+5] = c_2_5;
	C[(A_start_index+2)*C_width + B_start_index+6] = c_2_6;
	C[(A_start_index+2)*C_width + B_start_index+7] = c_2_7;
	C[(A_start_index+3)*C_width + B_start_index+0] = c_3_0;
	C[(A_start_index+3)*C_width + B_start_index+1] = c_3_1;
	C[(A_start_index+3)*C_width + B_start_index+2] = c_3_2;
	C[(A_start_index+3)*C_width + B_start_index+3] = c_3_3;
	C[(A_start_index+3)*C_width + B_start_index+4] = c_3_4;
	C[(A_start_index+3)*C_width + B_start_index+5] = c_3_5;
	C[(A_start_index+3)*C_width + B_start_index+6] = c_3_6;
	C[(A_start_index+3)*C_width + B_start_index+7] = c_3_7;
	C[(A_start_index+4)*C_width + B_start_index+0] = c_4_0;
	C[(A_start_index+4)*C_width + B_start_index+1] = c_4_1;
	C[(A_start_index+4)*C_width + B_start_index+2] = c_4_2;
	C[(A_start_index+4)*C_width + B_start_index+3] = c_4_3;
	C[(A_start_index+4)*C_width + B_start_index+4] = c_4_4;
	C[(A_start_index+4)*C_width + B_start_index+5] = c_4_5;
	C[(A_start_index+4)*C_width + B_start_index+6] = c_4_6;
	C[(A_start_index+4)*C_width + B_start_index+7] = c_4_7;
	C[(A_start_index+5)*C_width + B_start_index+0] = c_5_0;
	C[(A_start_index+5)*C_width + B_start_index+1] = c_5_1;
	C[(A_start_index+5)*C_width + B_start_index+2] = c_5_2;
	C[(A_start_index+5)*C_width + B_start_index+3] = c_5_3;
	C[(A_start_index+5)*C_width + B_start_index+4] = c_5_4;
	C[(A_start_index+5)*C_width + B_start_index+5] = c_5_5;
	C[(A_start_index+5)*C_width + B_start_index+6] = c_5_6;
	C[(A_start_index+5)*C_width + B_start_index+7] = c_5_7;
	C[(A_start_index+6)*C_width + B_start_index+0] = c_6_0;
	C[(A_start_index+6)*C_width + B_start_index+1] = c_6_1;
	C[(A_start_index+6)*C_width + B_start_index+2] = c_6_2;
	C[(A_start_index+6)*C_width + B_start_index+3] = c_6_3;
	C[(A_start_index+6)*C_width + B_start_index+4] = c_6_4;
	C[(A_start_index+6)*C_width + B_start_index+5] = c_6_5;
	C[(A_start_index+6)*C_width + B_start_index+6] = c_6_6;
	C[(A_start_index+6)*C_width + B_start_index+7] = c_6_7;
	C[(A_start_index+7)*C_width + B_start_index+0] = c_7_0;
	C[(A_start_index+7)*C_width + B_start_index+1] = c_7_1;
	C[(A_start_index+7)*C_width + B_start_index+2] = c_7_2;
	C[(A_start_index+7)*C_width + B_start_index+3] = c_7_3;
	C[(A_start_index+7)*C_width + B_start_index+4] = c_7_4;
	C[(A_start_index+7)*C_width + B_start_index+5] = c_7_5;
	C[(A_start_index+7)*C_width + B_start_index+6] = c_7_6;
	C[(A_start_index+7)*C_width + B_start_index+7] = c_7_7;


}
