#include "hip/hip_runtime.h"
/*Author: Swetha Varadarajan 6/10/2015
Adapted from CSU CS575 Spring 2011 
	which was based on code from the CUDA Programming Guide by David Newman
	Modified by Wim Bohm and David Newman
Changes made:
	1. Replaced cutil.h funcitons with normal timer functions adapted from CS475 assignments 
	2. Included the extern "C" in order to compile C code alongwith nvcc. 
	3. Replaced the Kernel and the header file name according to the functionality. 
	4. Kernel functionality changed from vector multiplication to max-add.	
	5. Time calculated is in seconds. So, the corresponding calculations are modified. (GOPS,GBYTES,TIME)
	6. FLOPS changed to OPS since this is an integer operation.
	7. Floats to Ints data type change for host and device vectors. 
*/

#include <stdio.h>
#include "intmaxaddF5K.h"

extern "C" {
#include "timer.h"
}

// Variables for host and device vectors.
int* h_A; 
int* h_B; 
int* h_C; 
int* d_A; 
int* d_B; 
int* d_C; 

// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{
    	int ValuesPerThread; // number of values per thread
   	int N; //Vector size
	int k; // no. of repeatitions
	int gridWidth = 60;
	int blockWidth = 1;

	// Parse arguments.
    	if(argc != 5){
    		 printf("Usage: %s ValuesPerThread Iterations\n", argv[0]);
     		 printf("ValuesPerThread is the number of values added by each thread.\n");
    		 printf("Total vector size is 128 * 60 * this value.\n");
    		 printf("Iterations is the number of repeatitions done by each thread.\n");
    		 exit(0);
   		 } 
	else 	{
     		 sscanf(argv[1], "%d", &ValuesPerThread);
      		 sscanf(argv[2], "%d", &k);
       		 sscanf(argv[3], "%d", &gridWidth);
      		 sscanf(argv[4], "%d", &blockWidth);
    		}      

	int size_A = blockWidth * ValuesPerThread;
	int size_B = gridWidth * ValuesPerThread;

	printf("Size of A: %d, Size of B: %d\n", size_A, size_B);
	
    	dim3 dimGrid(gridWidth);                    
    	dim3 dimBlock(blockWidth);                 

    	// Allocate input vectors h_A and h_B in host memory
    	h_A = (int*)malloc(size_A*sizeof(int));
    	if (h_A == 0) Cleanup(false);
    	h_B = (int*)malloc(size_B*sizeof(int));
    	if (h_B == 0) Cleanup(false);
    	h_C = (int*)malloc(size_A*sizeof(int)*size_B*sizeof(int));
    	if (h_C == 0) Cleanup(false);
	

   	// Allocate vectors in device memory.
   	hipError_t error;
    	error = hipMalloc((void**)&d_A, size_A*sizeof(int));
    	if (error != hipSuccess) Cleanup(false);
	
    	error = hipMalloc((void**)&d_B, size_B*sizeof(int));
    	if (error != hipSuccess) Cleanup(false);
	
   	error = hipMalloc((void**)&d_C, size_A*sizeof(int)*size_B*sizeof(int));
    	if (error != hipSuccess) Cleanup(false);


   	 // Initialize host vectors h_A and h_B
   	int i, j;
    	for(i=0; i <size_A; ++i){
    		 h_A[i] = (int)i;
   	 	}
    	for(i=0; i <size_B; ++i){
     		h_B[i] = (int)(N-i);   
    		}
	
	
    	// Copy host vectors h_A and h_B to device vectores d_A and d_B
	error = hipMemcpy(d_A, h_A, size_A*sizeof(int), hipMemcpyHostToDevice);
   	if (error != hipSuccess) Cleanup(false);
    	error = hipMemcpy(d_B, h_B, size_B*sizeof(int), hipMemcpyHostToDevice);
    	if (error != hipSuccess) Cleanup(false);
	

   	IntmaxaddF5<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);
    	error = hipGetLastError();

  	if (error != hipSuccess) {
		printf("W: %s\n", hipGetErrorString(error));
		Cleanup(false);
	}
    	hipDeviceSynchronize();



    	// Initialize timer
   	double time;
   	initialize_timer();
  	start_timer();
  
    	// Invoke kernel
    	IntmaxaddF5<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);
    	error = hipGetLastError();
    	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
		Cleanup(false);
	}


	// Compute elapsed time 
   	hipDeviceSynchronize();
	stop_timer();
   	time = elapsed_time();
        double nops = (double)size_A*(double)size_B*(double)k*(double)2 + (double)4*(double)k*(double)ValuesPerThread*(double)gridWidth*(double)blockWidth;
        float nopsPerSec = float(nops)/time;
    	float nGopsPerSec = nopsPerSec*1e-9;
	
	// Compute transfer rates.
    	int nBytes = size_A*sizeof(int) + size_B*sizeof(int) + size_A*sizeof(int)*size_B*sizeof(int); // 2N words in, N*N word out
    	float nBytesPerSec = (float)nBytes/time;
    	float nGBytesPerSec = nBytesPerSec*1e-9;

	// Report timing data.
    	printf( "Time: %f (s), Gops: %f, GBytesS: %f\n", time, nGopsPerSec, nGBytesPerSec);
     
    	// Copy result from device memory to host memory
   	error = hipMemcpy(h_C, d_C, size_A*sizeof(int)*size_B*sizeof(int), hipMemcpyDeviceToHost);
    	if (error != hipSuccess) Cleanup(false);

    	// Verify & report result
    	for (i = 0; i < size_A; ++i) {
    		for (j = 0; j < size_B; ++j) {
			int val = h_C[i*size_B+j];
			if (fabs(val - h_A[i]*h_B[j]) > 1e-5) {
				printf("Result error: i=%d, j=%d, expected %d, got %d\n", i, j, h_A[i]*h_B[j], val);
				break;
			}
		}
		if (j != size_B) {
			break;
		}
    	}
    	printf("Test %s \n", (i == size_A && j == size_B) ? "PASSED" : "FAILED");

	
    	Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    error = hipDeviceReset();
    checkCUDAError("tmp"); 
    if (!noError || error != hipSuccess)
        printf("error: %s cuda malloc or cuda thread exit failed \n", hipGetErrorString(hipGetLastError()));
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
		printf("Error");
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      exit(-1);
    }                         
}


