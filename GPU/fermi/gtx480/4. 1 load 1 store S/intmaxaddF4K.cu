
#include <hip/hip_runtime.h>
/*Author: Swetha Varadarajan 6/10/2015
 Adapted from Waruna Ranasinghe's summer 2014 work on kpdp micro-benchmarking 
 Code generated from intmazaddF4CG.c 
 Parameters list
size of A = 3264
    size of B = 1440
    gridDim = 240
    blockDim = 544
    Iterations k= 40000
   Values per thread x = 6
*/

__global__ void IntmaxaddF4(const int* A, const int* B, int* C, int x, int k)
{
	int B_start_index = (blockIdx.x*gridDim.y + blockIdx.y)*x;
	int A_start_index = (threadIdx.x*blockDim.y + threadIdx.y)*x;
	int C_width = x*gridDim.x*gridDim.y;


	int i,j,t;
	__shared__ int c[544][6][6];
	int a_0, a_1, a_2, a_3, a_4, a_5;
	int b_0, b_1, b_2, b_3, b_4, b_5;


	a_0 = A[A_start_index+0];
	a_1 = A[A_start_index+1];
	a_2 = A[A_start_index+2];
	a_3 = A[A_start_index+3];
	a_4 = A[A_start_index+4];
	a_5 = A[A_start_index+5];


	b_0 = B[B_start_index+0];
	b_1 = B[B_start_index+1];
	b_2 = B[B_start_index+2];
	b_3 = B[B_start_index+3];
	b_4 = B[B_start_index+4];
	b_5 = B[B_start_index+5];


	#pragma unroll
	for(i=0;i<x;i++)
	{
		for(j=0;j<x;j++)
		{
			c[threadIdx.x][i][j]=0;
		}
	}
	for (t = 0; t < k; t++) {
		c[threadIdx.x][0][0] = max(a_0,(c[threadIdx.x][0][0]+b_0));
		c[threadIdx.x][0][1] = max(a_1,(c[threadIdx.x][0][1]+b_1));
		c[threadIdx.x][0][2] = max(a_2,(c[threadIdx.x][0][2]+b_2));
		c[threadIdx.x][0][3] = max(a_3,(c[threadIdx.x][0][3]+b_3));
		c[threadIdx.x][0][4] = max(a_4,(c[threadIdx.x][0][4]+b_4));
		c[threadIdx.x][0][5] = max(a_5,(c[threadIdx.x][0][5]+b_5));
		c[threadIdx.x][1][0] = max(a_0,(c[threadIdx.x][1][0]+b_0));
		c[threadIdx.x][1][1] = max(a_1,(c[threadIdx.x][1][1]+b_1));
		c[threadIdx.x][1][2] = max(a_2,(c[threadIdx.x][1][2]+b_2));
		c[threadIdx.x][1][3] = max(a_3,(c[threadIdx.x][1][3]+b_3));
		c[threadIdx.x][1][4] = max(a_4,(c[threadIdx.x][1][4]+b_4));
		c[threadIdx.x][1][5] = max(a_5,(c[threadIdx.x][1][5]+b_5));
		c[threadIdx.x][2][0] = max(a_0,(c[threadIdx.x][2][0]+b_0));
		c[threadIdx.x][2][1] = max(a_1,(c[threadIdx.x][2][1]+b_1));
		c[threadIdx.x][2][2] = max(a_2,(c[threadIdx.x][2][2]+b_2));
		c[threadIdx.x][2][3] = max(a_3,(c[threadIdx.x][2][3]+b_3));
		c[threadIdx.x][2][4] = max(a_4,(c[threadIdx.x][2][4]+b_4));
		c[threadIdx.x][2][5] = max(a_5,(c[threadIdx.x][2][5]+b_5));
		c[threadIdx.x][3][0] = max(a_0,(c[threadIdx.x][3][0]+b_0));
		c[threadIdx.x][3][1] = max(a_1,(c[threadIdx.x][3][1]+b_1));
		c[threadIdx.x][3][2] = max(a_2,(c[threadIdx.x][3][2]+b_2));
		c[threadIdx.x][3][3] = max(a_3,(c[threadIdx.x][3][3]+b_3));
		c[threadIdx.x][3][4] = max(a_4,(c[threadIdx.x][3][4]+b_4));
		c[threadIdx.x][3][5] = max(a_5,(c[threadIdx.x][3][5]+b_5));
		c[threadIdx.x][4][0] = max(a_0,(c[threadIdx.x][4][0]+b_0));
		c[threadIdx.x][4][1] = max(a_1,(c[threadIdx.x][4][1]+b_1));
		c[threadIdx.x][4][2] = max(a_2,(c[threadIdx.x][4][2]+b_2));
		c[threadIdx.x][4][3] = max(a_3,(c[threadIdx.x][4][3]+b_3));
		c[threadIdx.x][4][4] = max(a_4,(c[threadIdx.x][4][4]+b_4));
		c[threadIdx.x][4][5] = max(a_5,(c[threadIdx.x][4][5]+b_5));
		c[threadIdx.x][5][0] = max(a_0,(c[threadIdx.x][5][0]+b_0));
		c[threadIdx.x][5][1] = max(a_1,(c[threadIdx.x][5][1]+b_1));
		c[threadIdx.x][5][2] = max(a_2,(c[threadIdx.x][5][2]+b_2));
		c[threadIdx.x][5][3] = max(a_3,(c[threadIdx.x][5][3]+b_3));
		c[threadIdx.x][5][4] = max(a_4,(c[threadIdx.x][5][4]+b_4));
		c[threadIdx.x][5][5] = max(a_5,(c[threadIdx.x][5][5]+b_5));


		a_0 = max(a_0,(b_0+c[threadIdx.x][0][0]));
		a_1 = max(a_1,(b_1+c[threadIdx.x][0][1]));
		a_2 = max(a_2,(b_2+c[threadIdx.x][0][2]));
		a_3 = max(a_3,(b_3+c[threadIdx.x][0][3]));
		a_4 = max(a_4,(b_4+c[threadIdx.x][0][4]));
		a_5 = max(a_5,(b_5+c[threadIdx.x][0][5]));


		b_0 =  max(b_0,(a_0+c[threadIdx.x][1][0]));
		b_1 =  max(b_1,(a_1+c[threadIdx.x][1][1]));
		b_2 =  max(b_2,(a_2+c[threadIdx.x][1][2]));
		b_3 =  max(b_3,(a_3+c[threadIdx.x][1][3]));
		b_4 =  max(b_4,(a_4+c[threadIdx.x][1][4]));
		b_5 =  max(b_5,(a_5+c[threadIdx.x][1][5]));


	}


	C[(A_start_index+0)*C_width + B_start_index+0] = c[threadIdx.x][0][0];
	C[(A_start_index+0)*C_width + B_start_index+1] = c[threadIdx.x][0][1];
	C[(A_start_index+0)*C_width + B_start_index+2] = c[threadIdx.x][0][2];
	C[(A_start_index+0)*C_width + B_start_index+3] = c[threadIdx.x][0][3];
	C[(A_start_index+0)*C_width + B_start_index+4] = c[threadIdx.x][0][4];
	C[(A_start_index+0)*C_width + B_start_index+5] = c[threadIdx.x][0][5];
	C[(A_start_index+1)*C_width + B_start_index+0] = c[threadIdx.x][1][0];
	C[(A_start_index+1)*C_width + B_start_index+1] = c[threadIdx.x][1][1];
	C[(A_start_index+1)*C_width + B_start_index+2] = c[threadIdx.x][1][2];
	C[(A_start_index+1)*C_width + B_start_index+3] = c[threadIdx.x][1][3];
	C[(A_start_index+1)*C_width + B_start_index+4] = c[threadIdx.x][1][4];
	C[(A_start_index+1)*C_width + B_start_index+5] = c[threadIdx.x][1][5];
	C[(A_start_index+2)*C_width + B_start_index+0] = c[threadIdx.x][2][0];
	C[(A_start_index+2)*C_width + B_start_index+1] = c[threadIdx.x][2][1];
	C[(A_start_index+2)*C_width + B_start_index+2] = c[threadIdx.x][2][2];
	C[(A_start_index+2)*C_width + B_start_index+3] = c[threadIdx.x][2][3];
	C[(A_start_index+2)*C_width + B_start_index+4] = c[threadIdx.x][2][4];
	C[(A_start_index+2)*C_width + B_start_index+5] = c[threadIdx.x][2][5];
	C[(A_start_index+3)*C_width + B_start_index+0] = c[threadIdx.x][3][0];
	C[(A_start_index+3)*C_width + B_start_index+1] = c[threadIdx.x][3][1];
	C[(A_start_index+3)*C_width + B_start_index+2] = c[threadIdx.x][3][2];
	C[(A_start_index+3)*C_width + B_start_index+3] = c[threadIdx.x][3][3];
	C[(A_start_index+3)*C_width + B_start_index+4] = c[threadIdx.x][3][4];
	C[(A_start_index+3)*C_width + B_start_index+5] = c[threadIdx.x][3][5];
	C[(A_start_index+4)*C_width + B_start_index+0] = c[threadIdx.x][4][0];
	C[(A_start_index+4)*C_width + B_start_index+1] = c[threadIdx.x][4][1];
	C[(A_start_index+4)*C_width + B_start_index+2] = c[threadIdx.x][4][2];
	C[(A_start_index+4)*C_width + B_start_index+3] = c[threadIdx.x][4][3];
	C[(A_start_index+4)*C_width + B_start_index+4] = c[threadIdx.x][4][4];
	C[(A_start_index+4)*C_width + B_start_index+5] = c[threadIdx.x][4][5];
	C[(A_start_index+5)*C_width + B_start_index+0] = c[threadIdx.x][5][0];
	C[(A_start_index+5)*C_width + B_start_index+1] = c[threadIdx.x][5][1];
	C[(A_start_index+5)*C_width + B_start_index+2] = c[threadIdx.x][5][2];
	C[(A_start_index+5)*C_width + B_start_index+3] = c[threadIdx.x][5][3];
	C[(A_start_index+5)*C_width + B_start_index+4] = c[threadIdx.x][5][4];
	C[(A_start_index+5)*C_width + B_start_index+5] = c[threadIdx.x][5][5];


}
