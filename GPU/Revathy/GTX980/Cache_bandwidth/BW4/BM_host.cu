#include "hip/hip_runtime.h"

/************************************

CPU Host code for BW Benchmark 4

 s += A[i]


*************************************/


#include <stdio.h>

// Variables for host and device vectors.
int* h_A; 
//int* h_B; 
int* h_C; 
int* d_A; 
//int* d_B; 
int* d_C; 

//Kernel declaration
__global__ void BWkernel (int *A, int *C, int x);

// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);


int main(int argc, char ** argv)
{

	int i, x, B, T, vectorLen;
	
	// Parse arguments.
    if(argc != 4){
     printf("Usage: %s ValuesPerThread NumBlocks ThreadsPerBlock\n", argv[0]);
     printf("ValuesPerThread is the footprint of each thread in a block.\n");
     printf("NumBlocks is the number of threadblocks to be launched.\n");
     printf("ThreadsPerBlock is the number of threads in each threadblock.\n");
     printf("Total vector size is ValuesPerThread*ThreadsPerBlock*NumBlocks\n");
     exit(0);
    } else {
      sscanf(argv[1], "%d", &x);
      sscanf(argv[2], "%d", &B);
      sscanf(argv[3], "%d", &T);
    }
    
    vectorLen = x * T * B;
    
    //printf("VectorLen %d\n", vectorLen);
    //printf("x %d\n", x);
    //printf("Blocks %d\n", B);
    //printf("Threads %d\n", T);
    
    //Allocate vectors on host
    int size = vectorLen*sizeof(int);
    
    h_A = (int*)malloc(size);
    if (h_A == 0) Cleanup(false);
    /*
    h_B = (int*)malloc(size);
    if (h_B == 0) Cleanup(false);
    */
    h_C = (int*)malloc(size);
    if (h_C == 0) Cleanup(false);
    
    //Allocate vectors on device
    hipError_t error;
    
    error = hipMalloc((void**)&d_A, size);
    if (error != hipSuccess) Cleanup(false);
    /*
    error = hipMalloc((void**)&d_B, size);
    if (error != hipSuccess) Cleanup(false);
    */
    error = hipMalloc((void**)&d_C, size);
    if (error != hipSuccess) Cleanup(false);
    
    
    // Initialize input vectors on host
    for(i=0; i <vectorLen; ++i){
     h_A[i] = i;
    }
    /*
    for(i=0; i <vectorLen; ++i){
     h_B[i] =(100-i);   
    }*/

    // Copy host vectors h_A and h_B to device vectores d_A and d_B
    error = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);
    /*
    error = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);
    */
    
    //Set L1 cache configuration for the kernel on device
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(BWkernel), hipFuncCachePreferL1);
    
    // Warm up
    BWkernel <<<B, T>>>(d_A,d_C, x);
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("W: %s\n", hipGetErrorString(error));
		Cleanup(false);
	}
    hipDeviceSynchronize();
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // Invoke kernel
    BWkernel <<<B, T>>>(d_A,d_C,100);
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
		Cleanup(false);
	}

    // Compute elapsed time 
    hipDeviceSynchronize();
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    
    // Compute integer operations per second.
    double nOps = (double) vectorLen;
    float nOpsPerSec = 1e3*nOps/time;
    float nGOpsPerSec = nOpsPerSec*1e-9;
    
    // Compute transfer rates.
    float nBytes = vectorLen*sizeof(int); // 2N integer inputs + N integer output
    float nBytesPerSec = 1e3*nBytes/time;
    float nGBytesPerSec = nBytesPerSec*1e-9;
    
    //Print results
    //printf("GOPS %f\n", nGOpsPerSec);
    //printf("GBytesS %f \n",nGBytesPerSec);

    printf("%d\t%d\t%d\t%d\t%f\t%f\n",vectorLen, x, B, T, nGOpsPerSec, nGBytesPerSec);
    
    /*    
    // Copy result from device memory to host memory
    error = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) Cleanup(false);

    // Verify & report result
    for (i = 0; i < vectorLen; ++i) {
		if (h_C[i] != 100) {
			break;
		}
    }
    printf("Test %s \n", (i == vectorLen) ? "PASSED" : "FAILED");
    */
    
    /*
    printf("Printing inputs: \n h_A  h_B  h_C\n");
    for(i=0; i<vectorLen; i++){
		printf("%3d  %3d  %3d  \n", h_A[i], h_B[i], h_C[i]);
	}
	*/
	// Clean up and exit.
    hipEventDestroy(start);
    hipEventDestroy(stop);
    Cleanup(true);
    
    return 0;
	
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    
    if (d_A)
        hipFree(d_A);
    /*
    if (d_B)
        hipFree(d_B);
    */
    if (d_C)
        hipFree(d_C);

    // Free host memory
    
    if (h_A)
        free(h_A);
    /*
    if (h_B)
        free(h_B);
    */
    if (h_C)
        free(h_C);
        
    error = hipDeviceReset();
   checkCUDAError("tmp"); 
    if (!noError || error != hipSuccess)
        printf("error: %s cuda malloc or cuda thread exit failed \n", hipGetErrorString(hipGetLastError()));
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
		printf("Error");
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      exit(-1);
    }                         
}
