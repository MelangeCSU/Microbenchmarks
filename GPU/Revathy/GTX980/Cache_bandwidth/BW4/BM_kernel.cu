
#include <hip/hip_runtime.h>
/***********************************
This kernel is for testing L1 and L2 cache Bandwidth

BW Benchmark :s += A[i]
size of A : 143032320
TILE_SIZE     : 1191936

************************************/

#define TILE_SIZE 1191936

__global__ void BWkernel (int *A, int *C, int x)
{
	 int tId = threadIdx.x;
	 int bId = blockIdx.x;
	 int bDim = blockDim.x;

	int start_Idx = bId*TILE_SIZE + tId;

	int i, s;
	

	s = 0;

	//Main loop
	#pragma unroll
	for(i=start_Idx; i<TILE_SIZE; i+=bDim)
	{
		s += A[i];
	
	}
	
	if(tId==0) C[bId] = s; 
}
