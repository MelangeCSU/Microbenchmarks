
#include <hip/hip_runtime.h>
/***********************************
This kernel is for testing L1 and L2 cache Bandwidth

BW Benchmark :s += A[i]*B[i]
TILE_SIZE     : 4096

************************************/

#define TILE_SIZE 4096

__global__ void BWkernel (int *A, int *B, int *C, int x)
{
	 int tId = threadIdx.x;
	 int bId = blockIdx.x;
	 int bDim = blockDim.x;

	int start_Idx = bId*TILE_SIZE + tId;

	int i, s;
	

	s = 0;

	//Main loop
	#pragma unroll
	for(i=start_Idx; i<TILE_SIZE; i+=bDim)
	{
		s += A[i] * B[i];
	
	}
	
	if(tId==0) C[bId] = s; 
}
