
#include <hip/hip_runtime.h>
/***********************************
BW Benchmark :s = A[i]
size of A,B : 393216
TILE_SIZE     : 12288

************************************/

#define TILE_SIZE 12288
#define SHARED_SIZE 12288

__global__ void BWkernel (int *C, int x)
{
	 int tId = threadIdx.x;
	 int bDim = blockDim.x*1;


	int i, t, s[1];
	
	__shared__ int s_A[SHARED_SIZE];

	//Main loop
	#pragma unroll 1
	for(t=0; t<4000000; t++)
	{
	#pragma unroll 1
	for(i=0; i<TILE_SIZE; i+=bDim)
	{
		s[0] += s_A[i+tId+0];
	
	}
	}
	
	//Store result to global memory	
if(tId<1)		C[tId] = s[tId];
	
}
