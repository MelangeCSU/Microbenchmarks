
#include <hip/hip_runtime.h>
/** size of A = 768
    size of B = 180
    gridDim = 60
    blockDim = 256
    k= 200000
    x = 3
**/

__global__ void CompareAddVectors(const int* A, const int* B, int* C, int x, int k)
{
	int size_A = x*blockDim.x;
	int B_start_index = (blockIdx.x*gridDim.y + blockIdx.y)*x;

	int t,i,j,temp;
	__shared__ int c[3][768]; 
	 __shared__ int a[768]; 
	int b_0, b_1, b_2;


	for (t = 0; t < x; t++) {
		temp = blockDim.x*t + threadIdx.x;
		a[temp] = A[temp];
	}


	for (i = 0; i < x; i++) {
		for (t = 0; t < x; t++) {
			c[i][blockDim.x*t + threadIdx.x] = 0;
		}
	}


	__syncthreads(); 

	b_0 = B[B_start_index+0];
	b_1 = B[B_start_index+1];
	b_2 = B[B_start_index+2];




	for (t = 0; t < k; t++) {
		#pragma unroll
		for (j = 0; j < x; j++) {
			temp = blockDim.x*j+threadIdx.x;
			c[0][temp] = max(a[temp]+b_0, c[0][temp]);
		}
		#pragma unroll
		for (j = 0; j < x; j++) {
			temp = blockDim.x*j+threadIdx.x;
			c[1][temp] = max(a[temp]+b_1, c[1][temp]);
		}
		#pragma unroll
		for (j = 0; j < x; j++) {
			temp = blockDim.x*j+threadIdx.x;
			c[2][temp] = max(a[temp]+b_2, c[2][temp]);
		}


		for (i = 0; i < x; i++) {
			temp = blockDim.x*i+threadIdx.x;
			a[temp] = max(a[temp]+10, a[temp]);
		}


		b_0 = max(b_0, b_0+10);
		b_1 = max(b_1, b_1+10);
		b_2 = max(b_2, b_2+10);


		__syncthreads();
	}


	for (i = 0; i < x; i++) {
		for (t = 0; t < x; t++) {
			temp = t*blockDim.x + threadIdx.x;
			C[(x*blockIdx.x+i)*size_A + temp] = c[i][temp];
		}
	}


}
