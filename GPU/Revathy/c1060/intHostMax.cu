#include "hip/hip_runtime.h"

//
/// File: intHostMax.cu
/// 
/// Host code for microbenchmarks to measure calibrate
/// GPU capability for integer max-plus operation.
///
/// Last Modified by: Revathy
/// Last Modified: 2015-05-07
/// 

// Includes
#include <stdio.h>
//#include <cutil.h>
#include "intMax.h"

#define    MAX(x,y)   ((x)>(y) ? (x) : (y))

// Variables for host and device vectors.
int* h_A; 
int* h_B; 
int* h_C; 
int* d_A; 
int* d_B; 
int* d_C; 

// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{
    int ValuesPerThread; // number of values per thread
    int N; //Vector size
    int k; // no. of repeatitions
    int gridWidth = 60;
    int blockWidth = 1;

	// Parse arguments.
    if(argc != 5){
     printf("Usage: %s ValuesPerThread Iterations\n", argv[0]);
     printf("ValuesPerThread is the number of values added by each thread.\n");
     printf("Total vector size is 128 * 60 * this value.\n");
     printf("Iterations is the number of repeatitions done by each thread.\n");
     exit(0);
    } else {
      sscanf(argv[1], "%d", &ValuesPerThread);
      sscanf(argv[2], "%d", &k);
      sscanf(argv[3], "%d", &gridWidth);
      sscanf(argv[4], "%d", &blockWidth);
    }      

	int size_A = blockWidth * ValuesPerThread;
	int size_B = gridWidth * ValuesPerThread;

	printf("Size of A: %d, Size of B: %d\n", size_A, size_B);

   // Determine the number of threads .
   // N is the total number of values to be in a vector
   //N = ValuesPerThread * gridWidth * blockWidth;

    dim3 dimGrid(gridWidth);                    
    dim3 dimBlock(blockWidth);                 

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size_A*sizeof(int));
    if (h_A == 0) Cleanup(false);
    h_B = (int*)malloc(size_B*sizeof(int));
    if (h_B == 0) Cleanup(false);
    h_C = (int*)malloc(size_A*sizeof(int)*size_B*sizeof(int));
    if (h_C == 0) Cleanup(false);
	

    // Allocate vectors in device memory.
    hipError_t error;
    error = hipMalloc((void**)&d_A, size_A*sizeof(int));
    if (error != hipSuccess) Cleanup(false);
    error = hipMalloc((void**)&d_B, size_B*sizeof(int));
    if (error != hipSuccess) Cleanup(false);
    error = hipMalloc((void**)&d_C, size_A*sizeof(int)*size_B*sizeof(int));
    if (error != hipSuccess) Cleanup(false);

    // Initialize host vectors h_A and h_B
    int i, j;
    for(i=0; i <size_A; ++i){
     h_A[i] = i;
    }
    for(i=0; i <size_B; ++i){
     h_B[i] =(N-i);   
    }

    // Copy host vectors h_A and h_B to device vectores d_A and d_B
    error = hipMemcpy(d_A, h_A, size_A*sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);
    error = hipMemcpy(d_B, h_B, size_B*sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);

    // Warm up
    CompareAddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("W: %s\n", hipGetErrorString(error));
		Cleanup(false);
	}
    hipDeviceSynchronize();

    // Initialize timer
    //unsigned int timer = 0;
    //cutCreateTimer(&timer);
    //cutStartTimer(timer);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // Invoke kernel
    CompareAddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
		Cleanup(false);
	}

    // Compute elapsed time 
    hipDeviceSynchronize();
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);

    // Compute integer operations per second.
    double nOps = (double)size_A*(double)size_B*(double)k*(double)2 + (double)2*(double)2*(double)k*(double)ValuesPerThread*(double)gridWidth*(double)blockWidth;
    float nOpsPerSec = 1e3*nOps/time;
    float nGOpsPerSec = nOpsPerSec*1e-9;
    // Compute transfer rates.
    float nBytes = size_A*sizeof(int) + size_B*sizeof(int) + size_A*sizeof(int)*size_B*sizeof(int); // 2N words in, N*N word out
    float nBytesPerSec = 1e3*nBytes/time;
    float nGBytesPerSec = nBytesPerSec*1e-9;

    // Report timing data.
    //printf( "Time: %f (ms), GOPS: %f, GBytesS: %f\n", time, nGOpsPerSec, nGBytesPerSec);

    printf("ValuesPerThread: %d \n",ValuesPerThread);
    printf("Iterations: %d \n", k);
    printf("TB: %d \n",gridWidth);
    printf("TPB: %d \n", blockWidth);
    printf("\n");
    printf("Time: %f (ms)\n",time);
    printf("GOPS: %f \n",nGOpsPerSec );
    printf("GBytesS: %f \n",nGBytesPerSec);
     
    // Copy result from device memory to host memory
    error = hipMemcpy(h_C, d_C, size_A*sizeof(int)*size_B*sizeof(int), hipMemcpyDeviceToHost);
    if (error != hipSuccess) Cleanup(false);

    // Verify & report result
    for (i = 0; i < size_A; ++i) {
    	for (j = 0; j < size_B; ++j) {
			int val = h_C[i*size_B+j];
			if (abs(val - MAX(h_A[i]+h_B[j],0)) > 0) {
				printf("Result error: i=%d, j=%d, expected %d, got %d\n", i, j, MAX(h_A[i]+h_B[j],0.0), val);
				break;
			}
		}
		if (j != size_B) {
			break;
		}
    }
    printf("Test %s \n", (i == size_A && j == size_B) ? "PASSED" : "FAILED");

	// Clean up and exit.
    hipEventDestroy(start);
    hipEventDestroy(stop);
    Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    error = hipDeviceReset();
   checkCUDAError("tmp"); 
    if (!noError || error != hipSuccess)
        printf("error: %s cuda malloc or cuda thread exit failed \n", hipGetErrorString(hipGetLastError()));
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
		printf("Error");
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      exit(-1);
    }                         
}


